#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */  

 
#include <cctype>
#include <algorithm>
#include <functional>
#include <numeric>
#include "cm.h"
#include "atof.h"
#include "compress.cu"
#include "sorts.cu"
#include "filter.h"
#include "callbacks.h"

#ifdef _WIN64
#define atoll(S) _atoi64(S)
#include <windows.h>
#else
#include <unistd.h>
#endif

using namespace std;

size_t total_count = 0, total_max;
clock_t tot;
unsigned int total_segments = 0;
unsigned int process_count;
size_t alloced_sz = 0;
bool fact_file_loaded = 1;
bool verbose;
bool interactive, ssd, delta, star;
void* d_v = nullptr;
void* s_v = nullptr;
queue<string> op_sort;
queue<string> op_presort;
queue<string> op_type;
bool op_case = 0;
queue<string> op_value;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<string> col_aliases;
map<string, map<string, col_data> > data_dict;
unordered_map<string, unordered_map<unsigned long long int, size_t> > char_hash;

map<string, char*> index_buffers;
map<string, char*> buffers;
map<string, size_t> buffer_sizes;
size_t total_buffer_size;
queue<string> buffer_names;

void* alloced_tmp;
bool alloced_switch = 0;
map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
map<string, unsigned int> cpy_bits;
map<string, long long int> cpy_init_val;

struct is_match
{
    __host__ __device__
    bool operator()(unsigned int x)
    {
        return x != 4294967295;
    }
};


struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON) || ((x-y) < -EPSILON);
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};


struct to_zero
{
    __host__ __device__
    bool operator()(const int_type x)
    {
        if(x == -1)
            return 0;
        else
            return 1;
    }
};



struct div_long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x, const float_type y)
    {
        return (float_type)x/y;
    }
};

char *mystrtok(char **m,char *s,const char c)
{
    char *p=s?s:*m;
    if( !*p )
        return 0;
    *m=strchr(p,c);
    if( *m )
        *(*m)++=0;
    else
        *m=p+strlen(p);
    return p;
}


void allocColumns(CudaSet* a, queue<string> fields);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt);
void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void write_compressed_char(string file_name, unsigned int index, size_t mCount);
size_t getFreeMem();
char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a, unsigned int segment);
size_t getTotalSystemMemory();
void process_error(int severity, string err);

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, queue<string> &references, queue<string> &references_names)
    : mColumnCount(0), mRecCount(0)
{
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, references, references_names);
    keep = false;
    source = 1;
    text_source = 1;
    fil_f = nullptr;
    fil_s = nullptr;
};

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name, unsigned int max)
    : mColumnCount(0),  mRecCount(0)
{
    maxRecs = max;
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
    keep = false;
    source = 1;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
};

CudaSet::CudaSet(const size_t RecordCount, const unsigned int ColumnCount)
{
    initialize(RecordCount, ColumnCount);
    keep = false;
    source = 0;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
};


CudaSet::CudaSet(queue<string> op_sel, const queue<string> op_sel_as)
{
    initialize(op_sel, op_sel_as);
    keep = false;
    source = 0;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
};

CudaSet::CudaSet(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    initialize(a,b, op_sel, op_sel_as);
    keep = false;
    source = 0;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
};


CudaSet::~CudaSet()
{
    free();
};


void CudaSet::allocColumnOnDevice(string colname, size_t RecordCount)
{
    if (type[colname] != 1 ) {
        d_columns_int[colname].resize(RecordCount);
    }
    else 
        d_columns_float[colname].resize(RecordCount);
};


void CudaSet::resize_join(size_t addRecs)
{
    mRecCount = mRecCount + addRecs;
    for(unsigned int i=0; i < columnNames.size(); i++) {
        if(type[columnNames[i]] != 1) {
            h_columns_int[columnNames[i]].resize(mRecCount);
        }
        else
            h_columns_float[columnNames[i]].resize(mRecCount);
    };
};


void CudaSet::resize(size_t addRecs)
{
    mRecCount = mRecCount + addRecs;
    for(unsigned int i=0; i < columnNames.size(); i++) {
        if(type[columnNames[i]] != 1) {
            h_columns_int[columnNames[i]].resize(mRecCount);
        }
        else {
            h_columns_float[columnNames[i]].resize(mRecCount);
        }
    };
};

void CudaSet::deAllocColumnOnDevice(string colname)
{
    if (type[colname] != 1 && !d_columns_int.empty()) {
        if(d_columns_int[colname].size() > 0) {
            d_columns_int[colname].resize(0);
            d_columns_int[colname].shrink_to_fit();
        };
    }
    else if (type[colname] == 1 && !d_columns_float.empty()) {
        if (d_columns_float[colname].size() > 0) {
            d_columns_float[colname].resize(0);
            d_columns_float[colname].shrink_to_fit();
        };
    };
};

void CudaSet::allocOnDevice(size_t RecordCount)
{
    for(unsigned int i=0; i < columnNames.size(); i++)
        allocColumnOnDevice(columnNames[i], RecordCount);
};

void CudaSet::deAllocOnDevice()
{
    for(unsigned int i=0; i < columnNames.size(); i++)
        deAllocColumnOnDevice(columnNames[i]);

    for (auto it=d_columns_int.begin(); it != d_columns_int.end(); ++it ) {
        if(it->second.size() > 0) {
            it->second.resize(0);
            it->second.shrink_to_fit();
        };
    };

    for (auto it=d_columns_float.begin(); it != d_columns_float.end(); ++it ) {
        if(it->second.size() > 0) {
            it->second.resize(0);
            it->second.shrink_to_fit();
        };
    };

    if(filtered) { // free the sources
        if(varNames.find(source_name) != varNames.end()) {
            varNames[source_name]->deAllocOnDevice();
        };
    };
};

void CudaSet::resizeDeviceColumn(size_t RecCount, string colname)
{
    if (type[colname] != 1) {
        d_columns_int[colname].resize(RecCount);
    }
    else 
        d_columns_float[colname].resize(RecCount);
};

void CudaSet::resizeDevice(size_t RecCount)
{
    for(unsigned int i=0; i < columnNames.size(); i++) {
        resizeDeviceColumn(RecCount, columnNames[i]);
    };
};

bool CudaSet::onDevice(string colname)
{
    if (type[colname] != 1) {
        if (!d_columns_int.empty() && d_columns_int[colname].size())
            return 1;
    }
    else 
        if (!d_columns_float.empty() && d_columns_float[colname].size())
            return 1;
    return 0;
}

CudaSet* CudaSet::copyDeviceStruct()
{

    CudaSet* a = new CudaSet(mRecCount, mColumnCount);
    a->not_compressed = not_compressed;
    a->segCount = segCount;
    a->maxRecs = maxRecs;
    a->ref_joins = ref_joins;
    a->ref_sets = ref_sets;
    a->ref_cols = ref_cols;
    a->columnNames = columnNames;
    a->cols = cols;
    a->type = type;
    a->char_size = char_size;
    a->decimal = decimal;

    for(unsigned int i=0; i < columnNames.size(); i++) {
        if(a->type[columnNames[i]] == 0) {
            a->d_columns_int[columnNames[i]] = thrust::device_vector<int_type>();
            a->h_columns_int[columnNames[i]] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
        }
        else if(a->type[columnNames[i]] == 1) {
            a->d_columns_float[columnNames[i]] = thrust::device_vector<float_type>();
            a->h_columns_float[columnNames[i]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
        }
        else {
            a->h_columns_char[columnNames[i]] = nullptr;
            a->d_columns_char[columnNames[i]] = nullptr;
        };
    };
    a->load_file_name = load_file_name;
    a->mRecCount = 0;
    return a;
}

int_type CudaSet::readSsdSegmentsFromFile(unsigned int segNum, string colname, size_t offset, thrust::host_vector<unsigned int>& prm_vh, CudaSet* dest)
{
    string f1 = load_file_name + "." + colname + "." + to_string(segNum);
    FILE* f = fopen(f1.c_str(), "rb" );
    if(!f) {
        cout << "Error opening " << f1 << " file " << endl;
        exit(0);
    };

    unsigned int cnt, bits;
    int_type lower_val;

    unsigned short int val_s_r[4096/2];
    char val_c_r[4096];
    unsigned int val_i_r[4096/4];
    unsigned long long int val_l_r[4096/8];
    unsigned int idx;
    bool idx_set = 0;

    fread(&cnt, 4, 1, f);
    fread(&lower_val, 8, 1, f);
    fseek(f, cnt - (8+4) + 32, SEEK_CUR);
    fread(&bits, 4, 1, f);
    //cout << "lower_val bits " << lower_val << " " << bits << endl;

    if(type[colname] == 0) {
        //cout << "lower_val bits " << lower_val << " " << bits << endl;

        for(unsigned int i = 0; i < prm_vh.size(); i++) {

            if(!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
                fseek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
                idx = prm_vh[i];
                idx_set = 1;

                if(bits == 8) {
                    fread(&val_c_r[0], 4096, 1, f);
                    dest->h_columns_int[colname][i + offset] = val_c_r[0];
                }
                else if(bits == 16) {
                    fread(&val_s_r, 4096, 1, f);
                    dest->h_columns_int[colname][i + offset] = val_s_r[0];
                }
                if(bits == 32) {
                    fread(&val_i_r, 4096, 1, f);
                    dest->h_columns_int[colname][i + offset] = val_i_r[0];
                }
                if(bits == 84) {
                    fread(&val_l_r, 4096, 1, f);
                    dest->h_columns_int[colname][i + offset] = val_l_r[0];
                }
            }
            else {
                if(bits == 8) {
                    dest->h_columns_int[colname][i + offset] = val_c_r[prm_vh[i]-idx];
                }
                else if(bits == 16) {
                    dest->h_columns_int[colname][i + offset] = val_s_r[prm_vh[i]-idx];
                }
                if(bits == 32) {
                    dest->h_columns_int[colname][i + offset] = val_i_r[prm_vh[i]-idx];
                }
                if(bits == 84) {
                    dest->h_columns_int[colname][i + offset] = val_l_r[prm_vh[i]-idx];
                }
            };
        };
    }
    else if(type[colname] == 1) {

        for(unsigned int i = 0; i < prm_vh.size(); i++) {
            if(!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
                fseek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
                idx = prm_vh[i];
                idx_set = 1;
                fread(val_c_r, 4096, 1, f);
                memcpy(&dest->h_columns_float[colname][i + offset], &val_c_r[0], bits/8);
            }
            else {
                memcpy(&dest->h_columns_float[colname][i + offset], &val_c_r[(prm_vh[i]-idx)*(bits/8)], bits/8);
            };
        };

    }
    else {
        //no strings in fact tables
    };
    fclose(f);
    return lower_val;
}

int_type CudaSet::readSsdSegmentsFromFileR(unsigned int segNum, string colname, thrust::host_vector<unsigned int>& prm_vh, thrust::host_vector<unsigned int>& dest)
{
    string f1 = load_file_name + "." + colname + "." + to_string(segNum);
    FILE* f = fopen(f1.c_str(), "rb" );
    if(!f) {
        cout << "Error opening " << f1 << " file " << endl;
        exit(0);
    };

    unsigned int cnt, bits;
    int_type lower_val;
    fread(&cnt, 4, 1, f);
    fread(&lower_val, 8, 1, f);
    fseek(f, cnt - (8+4) + 32, SEEK_CUR);
    fread(&bits, 4, 1, f);

    unsigned short int val_s_r[4096/2];
    char val_c_r[4096];
    unsigned int val_i_r[4096/4];
    unsigned long long int val_l_r[4096/8];
    unsigned int idx;
    bool idx_set = 0;

    for(unsigned int i = 0; i < prm_vh.size(); i++) {

        if(!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
            fseek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
            idx = prm_vh[i];
            idx_set = 1;

            if(bits == 8) {
                fread(val_c_r, 4096, 1, f);
                dest[i] = val_c_r[0];
            }
            else if(bits == 16) {
                fread(val_s_r, 4096, 1, f);
                dest[i] = val_s_r[0];
            }
            if(bits == 32) {
                fread(val_i_r, 4096, 1, f);
                dest[i] = val_i_r[0];
            }
            if(bits == 84) {
                fread(val_l_r, 4096, 1, f);
                dest[i] = val_l_r[0];
            }
        }
        else {
            if(bits == 8) {
                dest[i] = val_c_r[prm_vh[i]-idx];
            }
            else if(bits == 16) {
                dest[i] = val_s_r[prm_vh[i]-idx];
            }
            if(bits == 32) {
                dest[i] = val_i_r[prm_vh[i]-idx];
            }
            if(bits == 84) {
                dest[i] = val_l_r[prm_vh[i]-idx];
            }
        };
    };
    fclose(f);
    return lower_val;
}

std::clock_t tot_disk;

void CudaSet::readSegmentsFromFile(unsigned int segNum, string colname, size_t offset)
{
    string f1 = load_file_name + "." + colname + "." + to_string(segNum);
    if(type[colname] == 2)
        f1 = f1 + ".idx";

    std::clock_t start1 = std::clock();

    if(interactive) { //check if data are in buffers
        if(buffers.find(f1) == buffers.end()) { // add data to buffers
            FILE* f = fopen(f1.c_str(), "rb" );
            if(!f) {
                process_error(3, "Error opening " + string(f1) +" file " );
            };
            fseek(f, 0, SEEK_END);
            long fileSize = ftell(f);
            while(total_buffer_size + fileSize > getTotalSystemMemory() && !buffer_names.empty()) { //free some buffers
                //delete [] buffers[buffer_names.front()];
				hipHostFree(buffers[buffer_names.front()]);
                total_buffer_size = total_buffer_size - buffer_sizes[buffer_names.front()];
                buffer_sizes.erase(buffer_names.front());
                buffers.erase(buffer_names.front());
                buffer_names.pop();
            };
            fseek(f, 0, SEEK_SET);
			
			char* buff;
			hipHostAlloc((void**) &buff, fileSize,hipHostMallocDefault);
            fread(buff, fileSize, 1, f);
            fclose(f);
            buffers[f1] = buff;
            buffer_sizes[f1] = fileSize;
            buffer_names.push(f1);
            total_buffer_size = total_buffer_size + fileSize;
            buffer_names.push(f1);
            cout << "added buffer " << f1 << " " << fileSize << endl;
        };
        // get data from buffers
        if(type[colname] != 1) {
            unsigned int cnt = ((unsigned int*)buffers[f1])[0];
            if(cnt > h_columns_int[colname].size()/8 + 10)
                h_columns_int[colname].resize(cnt/8 + 10);
        }
        else {
            unsigned int cnt = ((unsigned int*)buffers[f1])[0];
            if(cnt > h_columns_float[colname].size()/8 + 10)
                h_columns_float[colname].resize(cnt/8 + 10);
        }
    }
    else {

        FILE* f = fopen(f1.c_str(), "rb" );
        if(!f) {
            cout << "Error opening " << f1 << " file " << endl;
            exit(0);
        };


        if(type[colname] != 1) {
            if(1 > h_columns_int[colname].size())
                h_columns_int[colname].resize(1);
            fread(h_columns_int[colname].data(), 4, 1, f);
            unsigned int cnt = ((unsigned int*)(h_columns_int[colname].data()))[0];
            if(cnt/8+10 > h_columns_int[colname].size())
                h_columns_int[colname].resize(cnt + 10);
            size_t rr = fread((unsigned int*)(h_columns_int[colname].data()) + 1, 1, cnt+52, f);
            if(rr != cnt+52) {
                char buf[1024];
                sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
                process_error(3, string(buf));
            };
        }
        else  {
            if(1 > h_columns_float[colname].size())
                h_columns_float[colname].resize(1);
            fread(h_columns_float[colname].data(), 4, 1, f);
            unsigned int cnt = ((unsigned int*)(h_columns_float[colname].data()))[0];
            if(cnt/8+10 > h_columns_float[colname].size())
                h_columns_float[colname].resize(cnt + 10);
            size_t rr = fread((unsigned int*)(h_columns_float[colname].data()) + 1, 1, cnt+52, f);
            if(rr != cnt+52) {
                char buf[1024];
                sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
                process_error(3, string(buf));
            };
        }
        fclose(f);
    };
    tot_disk =  tot_disk + (std::clock() - start1);
};

void CudaSet::CopyColumnToGpu(string colname,  unsigned int segment, size_t offset)
{

    if(not_compressed) 	{
        // calculate how many records we need to copy
        if(segment < segCount-1) {
            mRecCount = maxRecs;
        }
        else {
            mRecCount = hostRecCount - maxRecs*(segCount-1);
        };

        if(type[colname] != 1) {
            if(!alloced_switch)
                thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_columns_int[colname].begin() + offset);
            else {
                thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
                thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_col);
            };
        }
        else {
            if(!alloced_switch) {
                thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_columns_float[colname].begin() + offset);
            }
            else {
                thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
                thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_col);
            };
        }
    }
    else {

        readSegmentsFromFile(segment,colname, offset);
        if(!d_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if(!s_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

        string f1;
        if(type[colname] == 2) {
            f1 = load_file_name + "." + colname + "." + to_string(segment) + ".idx";
        }
        else {
            f1 = load_file_name + "." + colname + "." + to_string(segment);
        };

        if(type[colname] != 1) {
            if(!alloced_switch) {
                if(buffers.find(f1) == buffers.end()) {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + offset), h_columns_int[colname].data(), d_v, s_v, colname);
                }
                else {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + offset), buffers[f1], d_v, s_v, colname);
                };
            }
            else {
                if(buffers.find(f1) == buffers.end()) {
                    mRecCount = pfor_decompress(alloced_tmp, h_columns_int[colname].data(), d_v, s_v, colname);
                }
                else {
                    mRecCount = pfor_decompress(alloced_tmp, buffers[f1], d_v, s_v, colname);
                };
            };
        }
        else  {
            if(decimal[colname]) {
                if(!alloced_switch) {
                    if(buffers.find(f1) == buffers.end()) {
                        mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + offset) , h_columns_float[colname].data(), d_v, s_v, colname);
                    }
                    else {
                        mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + offset) , buffers[f1], d_v, s_v, colname);
                    };
					if(!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + offset));
						thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[colname].begin(), long_to_float());					
					};	
                }
                else {
                    if(buffers.find(f1) == buffers.end()) {
                        mRecCount = pfor_decompress(alloced_tmp, h_columns_float[colname].data(), d_v, s_v, colname);
                    }
                    else {
                        mRecCount = pfor_decompress(alloced_tmp, buffers[f1], d_v, s_v, colname);
                    };
					if(!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)alloced_tmp);
						thrust::device_ptr<float_type> d_col_float((float_type*)alloced_tmp);
						thrust::transform(d_col_int,d_col_int+mRecCount, d_col_float, long_to_float());
					};	
					//for(int i = 0; i < mRecCount;i++)
					//cout << "DECOMP " << (float_type)(d_col_int[i]) << " " << d_col_float[i] << endl;

                };
            }
            //else // uncompressed float
            // will have to fix it later so uncompressed data will be written by segments too
        }
    };
}



void CudaSet::CopyColumnToGpu(string colname) // copy all segments
{
    if(not_compressed) {
        if(type[colname] != 1)
            thrust::copy(h_columns_int[colname].begin(), h_columns_int[colname].begin() + mRecCount, d_columns_int[colname].begin());
        else
            thrust::copy(h_columns_float[colname].begin(), h_columns_float[colname].begin() + mRecCount, d_columns_float[colname].begin());
    }
    else {
        if(!d_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if(!s_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

        size_t cnt = 0;
        string f1;

        for(unsigned int i = 0; i < segCount; i++) {

            readSegmentsFromFile(i,colname, cnt);

            if(type[colname] == 2) {
                f1 = load_file_name + "." + colname + "." + to_string(i) + ".idx";
            }
            else {
                f1 = load_file_name + "." + colname + "." + to_string(i);
            };


            if(type[colname] == 0) {
                if(buffers.find(f1) == buffers.end()) {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + cnt), h_columns_int[colname].data(), d_v, s_v, colname);
                }
                else {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + cnt), buffers[f1], d_v, s_v, colname);
                };

            }
            else if(type[colname] == 1) {
                if(decimal[colname]) {
                    if(buffers.find(f1) == buffers.end()) {
                        mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt) , h_columns_float[colname].data(), d_v, s_v, colname);
                    }
                    else {
                        mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt) , buffers[f1], d_v, s_v, colname);
                    };
					if(!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt));
						thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[colname].begin() + cnt, long_to_float());
					};	
                }
                // else  uncompressed float
                // will have to fix it later so uncompressed data will be written by segments too
            };
            cnt = cnt + mRecCount;

            //totalRecs = totals + mRecCount;
        };

        mRecCount = cnt;
    };
}

void CudaSet::CopyColumnToHost(string colname, size_t offset, size_t RecCount)
{

    if(type[colname] != 1) {
		//cout << "copied " << colname << " " <<  RecCount << endl;
        thrust::copy(d_columns_int[colname].begin(), d_columns_int[colname].begin() + RecCount, h_columns_int[colname].begin() + offset);
		//cout << "to " << colname << " " << h_columns_int[colname][0] << " " << h_columns_int[colname][1] << endl;
		//cout << "to " << colname << " " << d_columns_int[colname][0] << " " << d_columns_int[colname][1] << endl;
	}
    else 
        thrust::copy(d_columns_float[colname].begin(), d_columns_float[colname].begin() + RecCount, h_columns_float[colname].begin() + offset);
}


void CudaSet::CopyColumnToHost(string colname)
{
    CopyColumnToHost(colname, 0, mRecCount);
}

void CudaSet::CopyToHost(size_t offset, size_t count)
{
    for(unsigned int i = 0; i < columnNames.size(); i++) {
        CopyColumnToHost(columnNames[i], offset, count);
    };
}

float_type* CudaSet::get_float_type_by_name(string name)
{
    return thrust::raw_pointer_cast(d_columns_float[name].data());
}

int_type* CudaSet::get_int_by_name(string name)
{
    return thrust::raw_pointer_cast(d_columns_int[name].data());
}

float_type* CudaSet::get_host_float_by_name(string name)
{
    return thrust::raw_pointer_cast(h_columns_float[name].data());
}

int_type* CudaSet::get_host_int_by_name(string name)
{
    return thrust::raw_pointer_cast(h_columns_int[name].data());
}



void CudaSet::GroupBy(stack<string> columnRef)
{
    if(grp.size() < mRecCount)
        grp.resize(mRecCount);	
	thrust::fill(grp.begin(), grp.begin()+mRecCount,0);
	if(scratch.size() < mRecCount)
		scratch.resize(mRecCount*sizeof(bool));
	thrust::device_ptr<bool> d_group((bool*)thrust::raw_pointer_cast(scratch.data()));	

    d_group[mRecCount-1] = 1;	
	unsigned int bits;	

    for(int i = 0; i < columnRef.size(); columnRef.pop()) {
        columnGroups.push(columnRef.top()); // save for future references

		
		if(cpy_bits.empty())
			bits = 0;
		else
			bits = cpy_bits[columnRef.top()];			
		
		if(bits == 8) {			
			if (type[columnRef.top()] != 1) {  // int_type
				thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
				thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned char>());			
			}
			else {
				thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(d_columns_float[columnRef.top()].data()));
				thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned char>());			
			};			
		}
		else if(bits == 16) {			
			if (type[columnRef.top()] != 1) {  // int_type
				thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
				thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned short int>());			
			}
			else {
				thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(d_columns_float[columnRef.top()].data()));
				thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned short int>());			
			};					
		}
		else if(bits == 32) {			
			if (type[columnRef.top()] != 1) {  // int_type
				thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
				thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned int>());			
			}
			else {
				thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(d_columns_float[columnRef.top()].data()));
				thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned int>());			
			};					
		}
		else {
			if (type[columnRef.top()] != 1) {  // int_type
				thrust::transform(d_columns_int[columnRef.top()].begin(), d_columns_int[columnRef.top()].begin() + mRecCount - 1,
					d_columns_int[columnRef.top()].begin()+1, d_group, thrust::not_equal_to<int_type>());
			}
			else {
				thrust::transform(d_columns_float[columnRef.top()].begin(), d_columns_float[columnRef.top()].begin() + mRecCount - 1,
                              d_columns_float[columnRef.top()].begin()+1, d_group, f_not_equal_to());
			};					
		}					

        thrust::transform(d_group, d_group+mRecCount, grp.begin(), grp.begin(), thrust::logical_or<bool>());
    };
    grp_count = thrust::count(grp.begin(), grp.begin()+mRecCount, 1);
};


void CudaSet::addDeviceColumn(int_type* col, string colname, size_t recCount)
{
    if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
        columnNames.push_back(colname);
        type[colname] = 0;
        d_columns_int[colname] = thrust::device_vector<int_type>(recCount);
        h_columns_int[colname] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >(recCount);
    }
    else {  // already exists, my need to resize it
        if(d_columns_int[colname].size() < recCount) {
            d_columns_int[colname].resize(recCount);
        };
		if(h_columns_int[colname].size() < recCount) {
            h_columns_int[colname].resize(recCount);
        };
    };
    // copy data to d columns
    thrust::device_ptr<int_type> d_col((int_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_int[colname].begin());
	thrust::copy(d_columns_int[colname].begin(), d_columns_int[colname].begin()+recCount, h_columns_int[colname].begin());
};

void CudaSet::addDeviceColumn(float_type* col, string colname, size_t recCount, bool is_decimal)
{
    if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
        columnNames.push_back(colname);
        type[colname] = 1;
        d_columns_float[colname] = thrust::device_vector<float_type>(recCount);
        h_columns_float[colname] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >(recCount);		
    }
    else {  // already exists, my need to resize it
        if(d_columns_float[colname].size() < recCount)
            d_columns_float[colname].resize(recCount);
        if(h_columns_float[colname].size() < recCount)
            h_columns_float[colname].resize(recCount);			
    };

    decimal[colname] = is_decimal;
    thrust::device_ptr<float_type> d_col((float_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_float[colname].begin());	
};

void CudaSet::compress(string file_name, size_t offset, unsigned int check_type, unsigned int check_val, size_t mCount)
{
    string str(file_name);
    thrust::device_vector<unsigned int> permutation;

    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));

    total_count = total_count + mCount;
    if (mCount > total_max && op_sort.empty()) {
        total_max = mCount;
    };

    if(!op_sort.empty()) { //sort the segment
        //copy the key columns to device
        queue<string> sf(op_sort);

        permutation.resize(mRecCount);
        thrust::sequence(permutation.begin(), permutation.begin() + mRecCount,0,1);
        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation.data());
        void* temp;

        CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount*max_char(this, sf)));

        string sort_type = "ASC";

        while(!sf.empty()) {

            if(type[sf.front()] != 2) {
                allocColumnOnDevice(sf.front(), maxRecs);
                CopyColumnToGpu(sf.front());
            };

            if (type[sf.front()] == 0)
                update_permutation(d_columns_int[sf.front()], raw_ptr, mRecCount, sort_type, (int_type*)temp, 64);
            else if (type[sf.front()] == 1)
                update_permutation(d_columns_float[sf.front()], raw_ptr, mRecCount, sort_type, (float_type*)temp, 64);
            else {
                thrust::host_vector<unsigned int> permutation_h = permutation;
                update_permutation_char_host(h_columns_char[sf.front()], permutation_h.data(), mRecCount, sort_type, (char*)temp, char_size[sf.front()]);
            };
            if(type[sf.front()] != 2)
                deAllocColumnOnDevice(sf.front());
            sf.pop();
        };
        hipFree(temp);
    };

    // here we need to check for partitions and if partition_count > 0 -> create partitions
    if(mCount < partition_count || partition_count == 0)
        partition_count = 1;
    unsigned int partition_recs = mCount/partition_count;

    if(!op_sort.empty()) {
        if(total_max < partition_recs)
            total_max = partition_recs;
    };

    total_segments++;
    unsigned int old_segments = total_segments;
    size_t new_offset;
    for(unsigned int i = 0; i < columnNames.size(); i++) {

        string colname = columnNames[i];

        str = file_name + "." + colname;
        curr_file = str;
        str += "." + to_string(total_segments-1);
        new_offset = 0;

        if(!op_sort.empty() && type[colname] != 2) {
            allocColumnOnDevice(colname, maxRecs);
            CopyColumnToGpu(colname);
        };		

        if(type[colname] == 0) {
            thrust::device_ptr<int_type> d_col((int_type*)d);
            if(!op_sort.empty()) {
                thrust::gather(permutation.begin(), permutation.end(), d_columns_int[colname].begin(), d_col);

                for(unsigned int p = 0; p < partition_count; p++) {
                    str = file_name + "." + colname;
                    curr_file = str;
                    str += "." + to_string(total_segments-1);
                    if (p < partition_count - 1) {
                        pfor_compress( (int_type*)d + new_offset, partition_recs*int_size, str, h_columns_int[colname], 0);
                    }
                    else {
                        pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*int_size, str, h_columns_int[colname], 0);
                    };
                    new_offset = new_offset + partition_recs;
                    total_segments++;
                };
            }
            else {
                thrust::copy(h_columns_int[colname].begin() + offset, h_columns_int[colname].begin() + offset + mCount, d_col);
                pfor_compress( d, mCount*int_size, str, h_columns_int[colname], 0);
            };
        }
        else if(type[colname] == 1) {
            if(decimal[colname]) {
                thrust::device_ptr<float_type> d_col((float_type*)d);
                if(!op_sort.empty()) {
                    thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());

                    for(unsigned int p = 0; p < partition_count; p++) {
                        str = file_name + "." + colname;
                        curr_file = str;
                        str += "." + to_string(total_segments-1);
                        if (p < partition_count - 1)
                            pfor_compress( (int_type*)d + new_offset, partition_recs*float_size, str, h_columns_float[colname], 1);
                        else
                            pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*float_size, str, h_columns_float[colname], 1);
                        new_offset = new_offset + partition_recs;
                        total_segments++;
                    };
                }
                else {
                    thrust::copy(h_columns_float[colname].begin() + offset, h_columns_float[colname].begin() + offset + mCount, d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());				
                    pfor_compress( d, mCount*float_size, str, h_columns_float[colname], 1);
                };
            }
            else { // do not compress -- float
                thrust::device_ptr<float_type> d_col((float_type*)d);
                if(!op_sort.empty()) {
                    thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
                    thrust::copy(d_col, d_col+mRecCount, h_columns_float[colname].begin());
                    for(unsigned int p = 0; p < partition_count; p++) {
                        str = file_name + "." + colname;
                        curr_file = str;
                        str += "." + to_string(total_segments-1);
                        unsigned int curr_cnt;
                        if (p < partition_count - 1)
                            curr_cnt = partition_recs;
                        else
                            curr_cnt = mCount - partition_recs*p;

                        fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
                        binary_file.write((char *)&curr_cnt, 4);
                        binary_file.write((char *)(h_columns_float[colname].data() + new_offset),curr_cnt*float_size);
                        new_offset = new_offset + partition_recs;
                        unsigned int comp_type = 3;
                        binary_file.write((char *)&comp_type, 4);
                        binary_file.close();
                    };
                }
                else {
                    fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
                    binary_file.write((char *)&mCount, 4);
                    binary_file.write((char *)(h_columns_float[colname].data() + offset),mCount*float_size);
                    unsigned int comp_type = 3;
                    binary_file.write((char *)&comp_type, 4);
                    binary_file.close();
                };
            };
        }
        else { //char
            if(!op_sort.empty()) {
                unsigned int*  h_permutation = new unsigned int[mRecCount];
                thrust::copy(permutation.begin(), permutation.end(), h_permutation);
                char* t = new char[char_size[colname]*mRecCount];
                apply_permutation_char_host(h_columns_char[colname], h_permutation, mRecCount, t, char_size[colname]);

                delete [] h_permutation;
                thrust::copy(t, t+ char_size[colname]*mRecCount, h_columns_char[colname]);
                delete [] t;
                for(unsigned int p = 0; p < partition_count; p++) {
                    str = file_name + "." + colname;
                    curr_file = str;
                    str += "." + to_string(total_segments-1);

                    if (p < partition_count - 1)
                        compress_char(str, colname, partition_recs, new_offset, total_segments-1);
                    else
                        compress_char(str, colname, mCount - partition_recs*p, new_offset, total_segments-1);
                    new_offset = new_offset + partition_recs;
                    total_segments++;
                };
            }
            else {
                compress_char(str, colname, mCount, offset, total_segments-1);
            };
        };

        if(type[colname] != 2)
            deAllocColumnOnDevice(colname);


        if((check_type == 1 && fact_file_loaded) || (check_type == 1 && check_val == 0)) {
            if(!op_sort.empty())
                writeHeader(file_name, colname, total_segments-1);
            else {
                writeHeader(file_name, colname, total_segments);
            };
        };

        total_segments = old_segments;
    };
    hipFree(d);

    if(!op_sort.empty()) {
        total_segments = (old_segments-1)+partition_count;
    };
    permutation.resize(0);
    permutation.shrink_to_fit();
}


void CudaSet::writeHeader(string file_name, string colname, unsigned int tot_segs) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";

    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&total_count, 8);
    binary_file.write((char *)&tot_segs, 4);
    binary_file.write((char *)&total_max, 4);
    binary_file.write((char *)&cnt_counts[ff], 4);
    binary_file.close();
};

void CudaSet::reWriteHeader(string file_name, string colname, unsigned int tot_segs, size_t newRecs, size_t maxRecs1) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";
    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&newRecs, 8);
    binary_file.write((char *)&tot_segs, 4);
    binary_file.write((char *)&maxRecs1, 4);
    binary_file.close();
};



void CudaSet::writeSortHeader(string file_name)
{
    string str(file_name);
    unsigned int idx;

    if(!op_sort.empty()) {
        str += ".sort";
        fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
        idx = (unsigned int)op_sort.size();
        binary_file.write((char *)&idx, 4);
        queue<string> os(op_sort);
        while(!os.empty()) {
            //idx = cols[columnNames[os.front()]];
            if(verbose)
                cout << "sorted on " << idx << endl;
            idx = os.front().size();
            binary_file.write((char *)&idx, 4);
            binary_file.write(os.front().data(), idx);
            os.pop();
        };
        binary_file.close();
    }
    else {
        str += ".sort";
        remove(str.c_str());
    };

    if(!op_presort.empty()) {
        str += ".presort";
        fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
        idx = (unsigned int)op_presort.size();
        binary_file.write((char *)&idx, 4);
        queue<string> os(op_presort);
        while(!os.empty()) {
            //idx = cols[columnNames[os.front()]];
            idx = os.front().size();
            binary_file.write((char *)&idx, 4);
            binary_file.write(os.front().data(), idx);
            os.pop();
        };
        binary_file.close();
    }
    else {
        str += ".presort";
        remove(str.c_str());
    };
}

using namespace mgpu;

void CudaSet::Display(unsigned int limit, bool binary, bool term)
{
#define MAXCOLS 128
#define MAXFIELDSIZE 1400

    //-- This should/will be converted to an array holding pointers of malloced sized structures--
    char    bigbuf[MAXCOLS * MAXFIELDSIZE];
    memset(bigbuf, 0, MAXCOLS * MAXFIELDSIZE);
    char    *fields[MAXCOLS];
    const   char *dcolumns[MAXCOLS];
    size_t  mCount;         // num records in play
    bool    print_all = 0;
    string  ss;
    int rows = 0;

    if(limit != 0 && limit < mRecCount)
        mCount = limit;
    else {
        mCount = mRecCount;
        print_all = 1;
    };

    cout << "mRecCount=" << mRecCount << " mcount = " << mCount << " term " << term <<  " limit=" << limit << " print_all=" << print_all << endl;

    unsigned int cc =0;
    unordered_map<string, FILE*> file_map;
    unordered_map<string, unsigned int> len_map;

    for(unsigned int i = 0; i < columnNames.size(); i++)
    {
        fields[cc] = &(bigbuf[cc*MAXFIELDSIZE]);                        // a hack to avoid malloc overheads     - refine later
        dcolumns[cc++] = columnNames[i].c_str();
    
		if(string_map.find(columnNames[i]) != string_map.end()) {
			auto s = string_map[columnNames[i]];
			auto pos = s.find_first_of(".");
			auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
			FILE *f;
			f = fopen(string_map[columnNames[i]].c_str(), "rb");
			file_map[string_map[columnNames[i]]] = f;
			len_map[string_map[columnNames[i]]] = len;
		};
    };

    // The goal here is to loop fast and avoid any double handling of outgoing data - pointers are good.
    if(not_compressed && prm_d.size() == 0) {
        for(unsigned int i=0; i < mCount; i++) {                            // for each record
            for(unsigned int j=0; j < columnNames.size(); j++) {                // for each col
                if (type[columnNames[j]] != 1) {
                    if(string_map.find(columnNames[j]) == string_map.end())
                        sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
                    else {
                        fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
                        fread(fields[j], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
                        fields[j][len_map[string_map[columnNames[j]]]] ='\0'; // zero terminate string
                    };
                }
                else
                    sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i] );
            };
            row_cb(mColumnCount, (char **)fields, (char **)dcolumns);
            rows++;
        };
    }
    else {
        queue<string> op_vx;
        for(unsigned int i = 0; i < columnNames.size(); i++)
            op_vx.push(columnNames[i]);


        if(prm_d.size() || source) {
            allocColumns(this, op_vx);
        };
        unsigned int curr_seg = 0;
        size_t cnt = 0;
        size_t curr_count, sum_printed = 0;
        resize(maxRecs);
        while(sum_printed < mCount || print_all) {

            if(prm_d.size() || source)  {                            // if host arrays are empty
                copyColumns(this, op_vx, curr_seg, cnt);
                size_t olRecs = mRecCount;
                mRecCount = olRecs;
                CopyToHost(0,mRecCount);
                if(sum_printed + mRecCount <= mCount || print_all)
                    curr_count = mRecCount;
                else
                    curr_count = mCount - sum_printed;
            }
            else
                curr_count = mCount;

            sum_printed = sum_printed + mRecCount;
            for(unsigned int i=0; i < curr_count; i++) {
                for(unsigned int j=0; j < columnNames.size(); j++) {
                    if (type[columnNames[j]] != 1) {
                        if(string_map.find(columnNames[j]) == string_map.end())
                            sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
                        else {
                            fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
                            fread(fields[j], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
                            fields[j][len_map[string_map[columnNames[j]]]] ='\0'; // zero terminate string
                        };
                    }
                    else
                        sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i] );
                };
                row_cb(mColumnCount, (char **)fields, (char**)dcolumns);
                rows++;
            };
            curr_seg++;
            if(curr_seg == segCount)
                print_all = 0;
        };
    };      // end else
    for(auto it = file_map.begin(); it != file_map.end(); it++)
		fclose(it->second);
}

void CudaSet::Store(const string file_name, const char* sep, const unsigned int limit, const bool binary, const bool term)
{
    if (mRecCount == 0 && binary == 1 && !term) { // write tails
        for(unsigned int j=0; j < columnNames.size(); j++) {
            writeHeader(file_name, columnNames[j], total_segments);
        };
        return;
    };

    size_t mCount;
    bool print_all = 0;

    if(limit != 0 && limit < mRecCount)
        mCount = limit;
    else {
        mCount = mRecCount;
        print_all = 1;
    };
    //cout << "mCount " << mCount << " " << mRecCount << endl;

    if(binary == 0) {

        unordered_map<string, FILE*> file_map;
        unordered_map<string, unsigned int> len_map;
        string bf;
        unsigned int max_len = 0;
        for(unsigned int j=0; j < columnNames.size(); j++) {
            if(string_map.find(columnNames[j]) != string_map.end()) {
                auto s = string_map[columnNames[j]];
                auto pos = s.find_first_of(".");
                auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
                if(len > max_len)
                    max_len = len;
                FILE *f;
                f = fopen(string_map[columnNames[j]].c_str(), "rb");
                file_map[string_map[columnNames[j]]] = f;
                len_map[string_map[columnNames[j]]] = len;
            };
        };
        bf.reserve(max_len);

        FILE *file_pr;
        if(!term) {
            file_pr = fopen(file_name.c_str(), "w");
            if (!file_pr)
                cout << "Could not open file " << file_name << endl;
        }
        else
            file_pr = stdout;

        if(not_compressed && prm_d.size() == 0) {
            for(unsigned int i=0; i < mCount; i++) {
                for(unsigned int j=0; j < columnNames.size(); j++) {
                    if (type[columnNames[j]] != 1 ) {
                        if(string_map.find(columnNames[j]) == string_map.end()) {
                            fprintf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
						}									
                        else {
                            //fprintf(file_pr, "%.*s", string_hash[columnNames[j]][h_columns_int[columnNames[j]][i]].size(), string_hash[columnNames[j]][h_columns_int[columnNames[j]][i]].c_str());
                            fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
                            fread(&bf[0], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
                            fprintf(file_pr, "%.*s", len_map[string_map[columnNames[j]]], bf.c_str());
                        };
                        fputs(sep, file_pr);
                    }
                    else {
                        fprintf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
                        fputs(sep, file_pr);
                    }
                };
                if (i != mCount -1 )
                    fputs("\n",file_pr);
            };
            if(!term)
                fclose(file_pr);
        }
        else {

            queue<string> op_vx;
            string ss;
            for(unsigned int j=0; j < columnNames.size(); j++)
                op_vx.push(columnNames[j]);

            if(prm_d.size() || source) {
                allocColumns(this, op_vx);
            };

            unsigned int curr_seg = 0;
            size_t cnt = 0;
            size_t curr_count, sum_printed = 0;
            mRecCount = 0;
            resize(maxRecs);

            while(sum_printed < mCount || print_all) {

                if(prm_d.size() || source)  {
                    copyColumns(this, op_vx, curr_seg, cnt);
                    if(curr_seg == 0) {
                        if(limit != 0 && limit < mRecCount) {
                            mCount = limit;
                            print_all = 0;
                        }
                        else {
                            mCount = mRecCount;
                            print_all = 1;
                        };

                    };

                    // if host arrays are empty
                    size_t olRecs = mRecCount;
                    mRecCount = olRecs;
                    CopyToHost(0,mRecCount);
                    //cout << "start " << sum_printed << " " <<  mRecCount << " " <<  mCount << endl;
                    if(sum_printed + mRecCount <= mCount || print_all) {
                        curr_count = mRecCount;
                    }
                    else {
                        curr_count = mCount - sum_printed;
                    };
                }
                else {
                    curr_count = mCount;
                };

                sum_printed = sum_printed + mRecCount;
                //cout << "sum printed " << sum_printed << " " << curr_count << " " << curr_seg << endl;

                for(unsigned int i=0; i < curr_count; i++) {
                    for(unsigned int j=0; j < columnNames.size(); j++) {
                        if (type[columnNames[j]] != 1) {
                            if(string_map.find(columnNames[j]) == string_map.end()) {
                                fprintf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
							}	
                            else {
                                fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
                                fread(&bf[0], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
                                fprintf(file_pr, "%.*s", len_map[string_map[columnNames[j]]], bf.c_str());
                            };
                            fputs(sep, file_pr);
                        }
                        else  {							
                            fprintf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
                            fputs(sep, file_pr);
                        };
                    };
                    if (i != mCount -1 && (curr_seg != segCount || i < curr_count))
                        fputs("\n",file_pr);
                };
                curr_seg++;
                if(curr_seg == segCount)
                    print_all = 0;
            };
            if(!term) {
                fclose(file_pr);
            };
        };
		for(auto it = file_map.begin(); it != file_map.end(); it++)
			fclose(it->second);		
    }
    else {
        //lets update the data dictionary
        for(unsigned int j=0; j < columnNames.size(); j++) {

            if(decimal[columnNames[j]] == 1)
                data_dict[file_name][columnNames[j]].col_type = 3;
            else
                data_dict[file_name][columnNames[j]].col_type = type[columnNames[j]];
            if(type[columnNames[j]] != 2)
                data_dict[file_name][columnNames[j]].col_length = 0;
            else
                data_dict[file_name][columnNames[j]].col_length = char_size[columnNames[j]];
        };
        save_dict = 1;


        if(text_source) {  //writing a binary file using a text file as a source

            // time to perform join checks on REFERENCES dataset segments
            //for(unsigned int i = 0; i< mColumnCount; i++) {

            for(unsigned int i=0; i < columnNames.size(); i++) {

                if(ref_sets.find(columnNames[i]) != ref_sets.end()) {

                    string f1 = file_name + "." + columnNames[i] + ".refs";
                    fstream f_file;
                    if(total_segments == 0) {
                        f_file.open(f1.c_str(), ios::out|ios::trunc|ios::binary);
                        unsigned int len = ref_sets[columnNames[i]].size();
                        f_file.write((char *)&len, 4);
                        f_file.write(ref_sets[columnNames[i]].c_str(), len);
                        len = ref_cols[columnNames[i]].size();
                        f_file.write((char *)&len, 4);
                        f_file.write(ref_cols[columnNames[i]].c_str(), len);
                    }
                    else {
                        f_file.open(f1.c_str(), ios::out|ios::app|ios::binary);
                    };

                    f1 = ref_sets[columnNames[i]] + "." + ref_cols[columnNames[i]] + ".header";
                    FILE* ff = fopen(f1.c_str(), "rb");
                    if(!ff) {
                        process_error(3, "Couldn't open file " + string(f1));
                    };
                    unsigned int ref_segCount, ref_maxRecs;
                    fread((char *)&ref_segCount, 4, 1, ff);
                    fread((char *)&ref_segCount, 4, 1, ff);
                    fread((char *)&ref_segCount, 4, 1, ff);
                    fread((char *)&ref_maxRecs, 4, 1, ff);
                    fclose(ff);
                    //cout << "CALC " << i << " " << columnNames[i] << " " << ref_sets[columnNames[i]] << " " << ref_cols[columnNames[i]] << " " << ref_segCount << " " << ref_maxRecs << endl;

                    CudaSet* a = new CudaSet(maxRecs, 1);
                    //a->h_columns_int[ref_cols[columnNames[i]]] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    a->h_columns_int[ref_cols[columnNames[i]]] = thrust::host_vector<int_type>();
                    a->d_columns_int[ref_cols[columnNames[i]]] = thrust::device_vector<int_type>(ref_maxRecs);
                    a->type[ref_cols[columnNames[i]]] = 0;
                    a->not_compressed = 0;
                    a->load_file_name = ref_sets[columnNames[i]];
                    a->cols[1] = ref_cols[columnNames[i]];
                    a->columnNames.push_back(ref_cols[columnNames[i]]);
                    MGPU_MEM(int) aIndicesDevice, bIndicesDevice;
                    size_t res_count;

                    if(!onDevice(columnNames[i])) {
                        allocColumnOnDevice(columnNames[i], maxRecs);
                    };
                    CopyColumnToGpu(columnNames[i]);
                    thrust::sort(d_columns_int[columnNames[i]].begin(), d_columns_int[columnNames[i]].begin() + mRecCount);

                    f_file.write((char *)&total_segments, 4);
                    f_file.write((char *)&ref_segCount, 4);
                    for(unsigned int z = 0; z < ref_segCount; z++) {

                        a->CopyColumnToGpu(ref_cols[columnNames[i]], z, 0);
                        thrust::sort(a->d_columns_int[ref_cols[columnNames[i]]].begin(), a->d_columns_int[ref_cols[columnNames[i]]].begin() + a->mRecCount);
                        // check if there is a join result
                        //cout << "join " << mRecCount << " " << a->mRecCount << " " << getFreeMem() << endl;
                        //cout << d_columns_int[columnNames[i]][0] << " " <<  d_columns_int[columnNames[i]][mRecCount-1] << " " << a->d_columns_int[ref_cols[columnNames[i]]][a->mRecCount-1]	<< " " <<  a->d_columns_int[ref_cols[columnNames[i]]][0] << endl;
                        if(d_columns_int[columnNames[i]][0] > a->d_columns_int[ref_cols[columnNames[i]]][a->mRecCount-1]	||
                                d_columns_int[columnNames[i]][mRecCount-1] < a->d_columns_int[ref_cols[columnNames[i]]][0]) {
                            res_count = 0;
                        }
                        else {
                            res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()), mRecCount,
                                        thrust::raw_pointer_cast(a->d_columns_int[ref_cols[columnNames[i]]].data()), a->mRecCount,
                                        &aIndicesDevice, &bIndicesDevice,
                                        mgpu::less<int_type>(), *context);
                        };
                        cout << "RES " << i << " " << total_segments << ":" << z << " " << res_count << endl;
                        f_file.write((char *)&z, 4);
                        f_file.write((char *)&res_count, 8);
                    };
                    f_file.close();
                    a->deAllocColumnOnDevice(ref_cols[columnNames[i]]);
                    a->free();
                };
            };
            compress(file_name, 0, 1, 0, mCount);
            for(unsigned int i = 0; i< columnNames.size(); i++)
                if(type[columnNames[i]] == 2)
                    deAllocColumnOnDevice(columnNames[i]);
        }
        else { //writing a binary file using a binary file as a source
            fact_file_loaded = 1;
            size_t offset = 0;

            if(!not_compressed) { // records are compressed, for example after filter op.
                //decompress to host
                queue<string> op_vx;
                for(unsigned int i = 0; i< columnNames.size(); i++) {
                    op_vx.push(columnNames[i]);
                };

                allocColumns(this, op_vx);
                size_t oldCnt = mRecCount;
                mRecCount = 0;
                resize(oldCnt);
                mRecCount = oldCnt;
                for(unsigned int i = 0; i < segCount; i++) {
                    size_t cnt = 0;
                    copyColumns(this, op_vx, i, cnt);
                    CopyToHost(0, mRecCount);
                    offset = offset + mRecCount;
                    compress(file_name, 0, 0, i - (segCount-1), mRecCount);
                };
            }
            else {
                // now we have decompressed records on the host
                //call setSegments and compress columns in every segment

                segCount = (mRecCount/process_count + 1);
                offset = 0;

                for(unsigned int z = 0; z < segCount; z++) {

                    if(z < segCount-1) {
                        if(mRecCount < process_count) {
                            mCount = mRecCount;
                        }
                        else {
                            mCount = process_count;
                        }
                    }
                    else {
                        mCount = mRecCount - (segCount-1)*process_count;
                    };
                    compress(file_name, offset, 0, z - (segCount-1), mCount);
                    offset = offset + mCount;
                };
            };
        };
    };
}


void CudaSet::compress_char(const string file_name, const string colname, const size_t mCount, const size_t offset, const unsigned int segment)
{
    unsigned int len = char_size[colname];
    unsigned long long int string_hash;

    string h_name, i_name, file_no_seg = file_name.substr(0, file_name.find_last_of("."));
    i_name = file_no_seg + "." + to_string(segment) + ".idx";
    h_name = file_no_seg + "." + to_string(segment) + ".hash";
    fstream b_file_str, loc_hashes;

    fstream binary_file_h(h_name.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file_h.write((char *)&mCount, 4);

    if(segment == 0) {
        b_file_str.open(file_no_seg.c_str(),ios::out|ios::binary|ios::trunc);
    }
    else {
        b_file_str.open(file_no_seg.c_str(),ios::out|ios::binary|ios::app);
    };

    if(h_columns_int.find(colname) == h_columns_int.end())
        h_columns_int[colname] = thrust::host_vector<int_type >(mCount);
    if(d_columns_int.find(colname) == d_columns_int.end())
        d_columns_int[colname] = thrust::device_vector<int_type >(mCount);
		
    for (unsigned int i = 0 ; i < mCount; i++) {
        string_hash = MurmurHash64A(h_columns_char[colname] + (i+offset)*len, len, hash_seed)/2;
        binary_file_h.write((char *)&string_hash, 8);
        if(char_hash[colname].find(string_hash) == char_hash[colname].end()) {
			auto cnt = char_hash[colname].size();
            char_hash[colname][string_hash] = cnt;
            b_file_str.write((char *)h_columns_char[colname] + (i+offset)*len, len);
            h_columns_int[colname][i] = cnt;
        }
        else {
            h_columns_int[colname][i] = char_hash[colname][string_hash];
        };

    };

    thrust::device_vector<int_type> d_col(mCount);
    thrust::copy(h_columns_int[colname].begin(), h_columns_int[colname].begin() + mCount, d_col.begin());
    pfor_compress(thrust::raw_pointer_cast(d_col.data()), mCount*int_size, i_name, h_columns_int[colname], 0);

    binary_file_h.close();
    b_file_str.close();
};

void CudaSet::compress_int(const string file_name, const string colname, const size_t mCount)
{
    std::vector<unsigned int> dict_val;
    unsigned int bits_encoded;
    set<int_type> dict_s;
    map<int_type, unsigned int> d_ordered;

    for (unsigned int i = 0 ; i < mCount; i++) {
        int_type f = h_columns_int[colname][i];
        dict_s.insert(f);
    };

    unsigned int i = 0;
    for (auto it = dict_s.begin(); it != dict_s.end(); it++) {
        d_ordered[*it] = i++;
    };

    for (unsigned int i = 0 ; i < mCount; i++) {
        int_type f = h_columns_int[colname][i];
        dict_val.push_back(d_ordered[f]);
    };

    bits_encoded = (unsigned int)ceil(log2(double(d_ordered.size()+1)));
    //cout << "bits " << bits_encoded << endl;

    unsigned int sz = (unsigned int)d_ordered.size();
    // write to a file
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&sz, 4);

    for (auto it = d_ordered.begin(); it != d_ordered.end(); it++) {
        binary_file.write((char*)(&(it->first)), int_size);
    };

    unsigned int fit_count = 64/bits_encoded;
    unsigned long long int val = 0;
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&bits_encoded, 4);
    unsigned int curr_cnt = 1;
    unsigned int vals_count = (unsigned int)dict_val.size()/fit_count;
    if(!vals_count || dict_val.size()%fit_count)
        vals_count++;
    binary_file.write((char *)&vals_count, 4);
    unsigned int real_count = (unsigned int)dict_val.size();
    binary_file.write((char *)&real_count, 4);

    for(unsigned int i = 0; i < dict_val.size(); i++) {

        val = val | dict_val[i];

        if(curr_cnt < fit_count)
            val = val << bits_encoded;

        if( (curr_cnt == fit_count) || (i == (dict_val.size() - 1)) ) {
            if (curr_cnt < fit_count) {
                val = val << ((fit_count-curr_cnt)-1)*bits_encoded;
            };
            curr_cnt = 1;
            binary_file.write((char *)&val, int_size);
            val = 0;
        }
        else
            curr_cnt = curr_cnt + 1;
    };
    binary_file.close();
};




bool CudaSet::LoadBigFile(FILE* file_p)
{
    char line[2000];
    unsigned int current_column, count = 0;
    string colname;
    char *p,*t;
    const char* sep = separator.c_str();
    unsigned int maxx = cols.rbegin()->first;
	map<unsigned int, string>::iterator it;

    //clear the varchars
    //for(auto it=columnNames.begin(); it!=columnNames.end();it++) {
    for(unsigned int i = 0; i < mColumnCount; i++) {
        if(type[columnNames[i]] == 2) {
			if(!h_columns_char[columnNames[i]])
				h_columns_char[columnNames[i]] = new char[maxRecs*char_size[columnNames[i]]];
            memset(h_columns_char[columnNames[i]], 0, maxRecs*char_size[columnNames[i]]);
        };
    };
	
	vector<int> types;
	types.push_back(0);
	for(int i = 0; i < maxx; i++) {
		auto iter = cols.find(i+1);
		if(iter != cols.end())
			types.push_back(type[iter->second]);
		else	
			types.push_back(0);
	};
		
	
    while (count < process_count && fgets(line, 2000, file_p)) {
        strtok(line, "\n");
        current_column = 0;

        for(t=mystrtok(&p,line,*sep); t && current_column < maxx; t=mystrtok(&p,0,*sep)) {
            current_column++;
			it = cols.find(current_column);
            if(it == cols.end()) {
                continue;
            };

			switch(types[current_column]) {
			case 0 :            		
                if (strchr(t,'-') && t[0] != '-') { // handling possible dates
                    strncpy(t+4,t+5,2);
                    strncpy(t+6,t+8,2);
                    t[8] = '\0';
                    (h_columns_int[it->second])[count] = atoll(t);
                }
                else if (strchr(t,'/')) { // 4/30/2014
                    string s(t);
                    size_t pos1 = s.find_first_of("/",0);
                    size_t pos2 = s.find_first_of("/",pos1+1);
                    string month = s.substr(0,pos1);
                    if(month.length() == 1)
                        month = "0" + month;
                    string day = s.substr(pos1+1,pos2-pos1-1);
                    if(day.length() == 1)
                        day = "0" + day;
                    string s2 = s.substr(pos2+1, string::npos) + month + day;
                    (h_columns_int[it->second])[count] = atoll(s2.c_str());
                }
                else {
				
                    (h_columns_int[it->second])[count] = atoll(t);
                };				
				break;
            case 1 :
                (h_columns_float[it->second])[count] = atoff(t);
				break;            
            default :  //char
                strcpy(h_columns_char[it->second] + count*char_size[it->second], t);
				break;
            };			
        };
        count++;
    };
	
    mRecCount = count;
    if(count < process_count)  {
        fclose(file_p);
        return 1;
    }
    else
        return 0;
};


void CudaSet::free()  {
    for(unsigned int i = 0; i < columnNames.size(); i++ ) {
		if(type[columnNames[i]] == 0 ) {
			h_columns_int[columnNames[i]].resize(0);
			h_columns_int[columnNames[i]].shrink_to_fit();
		}
		else {
			h_columns_float[columnNames[i]].resize(0);
			h_columns_float[columnNames[i]].shrink_to_fit();
		};
    };
    prm_d.resize(0);
    prm_d.shrink_to_fit();
    deAllocOnDevice();
};

void alloc_pool(unsigned int maxRecs) {
	void* temp;
	CUDA_SAFE_CALL(hipMalloc((void **) &temp, 8*maxRecs));
	alloced_mem.push_back(temp);		
};	



bool* CudaSet::logical_and(bool* column1, bool* column2)
{
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}


bool* CudaSet::logical_or(bool* column1, bool* column2)
{
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}



bool* CudaSet::compare(int_type s, int_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if(d>s) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if(d<s) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if(d>=s) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if(d<=s) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if(d==s) res = 1;
        else res = 0;
    else // !=
        if(d!=s) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
};


bool* CudaSet::compare(float_type s, float_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if ((d-s) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if ((s-d) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
        else res = 0;
    else // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
}


bool* CudaSet::compare(int_type* column1, int_type d, int_type op_type)
{
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);

    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);

}

bool* CudaSet::compare(float_type* column1, float_type d, int_type op_type)
{
    thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr(column1);

    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
    else  // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

    return thrust::raw_pointer_cast(res);
}


bool* CudaSet::compare(int_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, float_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    return thrust::raw_pointer_cast(temp);

}


bool* CudaSet::compare(float_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr(column2);
    thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    thrust::device_free(dev_ptr2);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(int_type* column1, float_type* column2, string op_type, int reverse)
{
	if(alloced_mem.empty()) {								
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());								
    //thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation
    thrust::device_ptr<float_type> dev_ptr1(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);

}




int_type* CudaSet::op(int_type* column1, int_type* column2, string op_type, int reverse)
{
	if(alloced_mem.empty()) {								
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<int_type> temp((int_type*)alloced_mem.back());								
    //thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    }
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);

}

float_type* CudaSet::op(float_type* column1, float_type* column2, string op_type, int reverse)
{
	if(alloced_mem.empty()) {								
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());								
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    };
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type d, string op_type, int reverse)
{
	if(alloced_mem.empty()) {								
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<int_type> temp((int_type*)alloced_mem.back());								
    //thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);
    thrust::device_ptr<int_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<int_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    };
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(int_type* column1, float_type d, string op_type, int reverse)
{
	if(alloced_mem.empty()) {								
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());								
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr(column1);
    thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };
    thrust::device_free(dev_ptr1);
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(float_type* column1, float_type d, string op_type,int reverse)
{
	if(alloced_mem.empty()) {								
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());								
    thrust::device_ptr<float_type> dev_ptr1(column1);
	
    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
    }
    else	{
        if (op_type.compare("MUL") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    };
	alloced_mem.pop_back();
    return (float_type*)thrust::raw_pointer_cast(temp);
}

char CudaSet::loadIndex(const string index_name, const unsigned int segment)
{
    FILE* f;
    unsigned int bits_encoded, fit_count, vals_count, sz, real_count;
    void* d_str;
    string f1 = index_name + "." + to_string(segment);
    char res;

    if(interactive) {
        if(index_buffers.find(f1) == index_buffers.end()) {
            f = fopen (f1.c_str(), "rb" );
            fseek(f, 0, SEEK_END);
            long fileSize = ftell(f);
            char* buff;
            hipHostAlloc(&buff, fileSize, hipHostMallocDefault);

            fseek(f, 0, SEEK_SET);
            fread(buff, fileSize, 1, f);
            fclose(f);
            index_buffers[f1] = buff;
        };
        sz = ((unsigned int*)index_buffers[f1])[0];

        idx_dictionary_int[index_name].clear();
        for(unsigned int i = 0; i < sz; i++) {
            idx_dictionary_int[index_name][((int_type*)(index_buffers[f1]+4+8*i))[0]] = i;
        };
        vals_count = ((unsigned int*)(index_buffers[f1]+4 +8*sz))[2];
        real_count = ((unsigned int*)(index_buffers[f1]+4 +8*sz))[3];
        mRecCount = real_count;
        res = (index_buffers[f1]+4 +8*sz + (vals_count+2)*int_size)[0];
        hipMalloc((void **) &d_str, (vals_count+2)*int_size);
        hipMemcpy( d_str, (void *) &((index_buffers[f1]+4 +8*sz)[0]), (vals_count+2)*int_size, hipMemcpyHostToDevice);

        if(idx_vals.count(index_name))
            hipFree(idx_vals[index_name]);
        idx_vals[index_name] = (unsigned long long int*)d_str;

    }
    else {
        f = fopen (f1.c_str(), "rb" );
        fread(&sz, 4, 1, f);
        int_type* d_array = new int_type[sz];
        idx_dictionary_int[index_name].clear();
        fread((void*)d_array, sz*int_size, 1, f);
        for(unsigned int i = 0; i < sz; i++) {
            idx_dictionary_int[index_name][d_array[i]] = i;
            //cout << index_name  << " " << d_array[i] << " " << i << endl;
        };
        delete [] d_array;

        fread(&fit_count, 4, 1, f);
        fread(&bits_encoded, 4, 1, f);
        fread(&vals_count, 4, 1, f);
        fread(&real_count, 4, 1, f);
        mRecCount = real_count;

        unsigned long long int* int_array = new unsigned long long int[vals_count+2];
        fseek ( f , -16 , SEEK_CUR );
        fread((void*)int_array, 1, vals_count*8 + 16, f);
        fread(&res, 1, 1, f);
        fclose(f);
        void* d_str;
        hipMalloc((void **) &d_str, (vals_count+2)*int_size);
        hipMemcpy( d_str, (void *) int_array, (vals_count+2)*int_size, hipMemcpyHostToDevice);
        if(idx_vals.count(index_name))
            hipFree(idx_vals[index_name]);
        idx_vals[index_name] = (unsigned long long int*)d_str;
    }
    return res;
}



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name) // compressed data for DIM tables
{
    mColumnCount = (unsigned int)nameRef.size();
    FILE* f;
    string f1;
    unsigned int cnt;
    char buffer[4000];
    string str;
    not_compressed = 0;
    mRecCount = Recs;
    hostRecCount = Recs;
    totalRecs = Recs;
    load_file_name = file_name;

    f1 = file_name + ".sort";
    f = fopen (f1.c_str() , "rb" );
    if(f) {
        unsigned int sz, idx;
        fread((char *)&sz, 4, 1, f);
        for(unsigned int j = 0; j < sz; j++) {
            fread((char *)&idx, 4, 1, f);
            fread(buffer, idx, 1, f);
            str.assign(buffer, idx);
            sorted_fields.push(str);
            if(verbose)
                cout << "segment sorted on " << str << endl;
        };
        fclose(f);
    };

    f1 = file_name + ".presort";
    f = fopen (f1.c_str() , "rb" );
    if(f) {
        unsigned int sz, idx;
        fread((char *)&sz, 4, 1, f);
        for(unsigned int j = 0; j < sz; j++) {
            fread((char *)&idx, 4, 1, f);
            fread(buffer, idx, 1, f);
            str.assign(buffer, idx);
            presorted_fields.push(str);
            if(verbose)
                cout << "presorted on " << str << endl;
        };
        fclose(f);
    };

    tmp_table = 0;
    filtered = 0;

    for(unsigned int i=0; i < mColumnCount; i++) {

        //f1 = file_name + "." + nameRef.front() + ".0";
        //f = fopen (f1.c_str() , "rb" );
        //fread((char *)&bytes, 4, 1, f); //need to read metadata such as type and length
        //fclose(f);

        columnNames.push_back(nameRef.front());
        cols[colsRef.front()] = nameRef.front();

        if (((typeRef.front()).compare("decimal") == 0) || ((typeRef.front()).compare("int") == 0)) {
            f1 = file_name + "." + nameRef.front() + ".0";
            f = fopen (f1.c_str() , "rb" );
            if(!f) {
                cout << "Couldn't find field " << nameRef.front() << endl;
                exit(0);
            };
            for(unsigned int j = 0; j < 6; j++)
                fread((char *)&cnt, 4, 1, f);
            fclose(f);
            compTypes[nameRef.front()] = cnt;
        };

        //check the references
        f1 = file_name + "." + nameRef.front() + ".refs";
        f = fopen (f1.c_str() , "rb" );
        if(f) {
            unsigned int len;
            fread(&len, 4, 1, f);
            char* array = new char[len+1];			
			memset(array, 0, len+1);
            fread((void*)array, len, 1, f);
			string s(array);
            ref_sets[nameRef.front()] = s;
            delete [] array;
            unsigned int segs, seg_num, curr_seg;
            size_t res_count;
            fread(&len, 4, 1, f);
            char* array1 = new char[len+1];
			memset(array1, 0, len+1);
            fread((void*)array1, len, 1, f);
			string s1(array1);
            ref_cols[nameRef.front()] = s1;
            delete [] array1;

            unsigned int bytes_read = fread((void*)&curr_seg, 4, 1, f);

            while(bytes_read == 1) {
                fread((void*)&segs, 4, 1, f); //ref seg count
                //cout << "for " << i << " read " << array << " and " << z << " " << segs << endl;

                for(unsigned int j = 0; j < segs; j++) {
                    fread((void*)&seg_num, 4, 1, f);
                    fread((void*)&res_count, 8, 1, f);
                    //cout << "curr_seg " << curr_seg << " " << seg_num << " " << res_count << endl;
                    if(res_count)
                        ref_joins[columnNames[i]][curr_seg].insert(seg_num);
                    else
                        ref_joins[columnNames[i]][curr_seg].insert(std::numeric_limits<unsigned int>::max());
                };
                bytes_read = fread((void*)&curr_seg, 4, 1, f);
            };
            fclose(f);
        };

        if ((typeRef.front()).compare("int") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 0;
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            //h_columns_int[nameRef.front()] = thrust::host_vector<int_type >();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 0;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            //h_columns_float[nameRef.front()] = thrust::host_vector<float_type>();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type >();
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 1;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            //h_columns_float[nameRef.front()] = thrust::host_vector<float_type>();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
        }
        else {
            type[nameRef.front()] = 2;
            decimal[nameRef.front()] = 0;
            h_columns_char[nameRef.front()] = nullptr;
            d_columns_char[nameRef.front()] = nullptr;
            char_size[nameRef.front()] = sizeRef.front();
            string_map[nameRef.front()] = file_name + "." + nameRef.front();
        };

        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    };

};



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, queue<string> &references, queue<string> &references_names)
{
    mColumnCount = (unsigned int)nameRef.size();
    tmp_table = 0;
    filtered = 0;
    mRecCount = Recs;
    hostRecCount = Recs;
    segCount = 1;

    for(unsigned int i=0; i < mColumnCount; i++) {

        columnNames.push_back(nameRef.front());
        cols[colsRef.front()] = nameRef.front();

        if ((typeRef.front()).compare("int") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 0;
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            //h_columns_int[nameRef.front()] = thrust::host_vector<int_type>();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 0;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            //h_columns_float[nameRef.front()] = thrust::host_vector<float_type>();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 1;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            //h_columns_float[nameRef.front()] = thrust::host_vector<float_type>();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
        }

        else {
            type[nameRef.front()] = 2;
            decimal[nameRef.front()] = 0;
            h_columns_char[nameRef.front()] = nullptr;
            d_columns_char[nameRef.front()] = nullptr;
            char_size[nameRef.front()] = sizeRef.front();
        };

        if(!references.front().empty()) {
            ref_sets[nameRef.front()] = references.front();
            ref_cols[nameRef.front()] = references_names.front();
        };
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
        references.pop();
        references_names.pop();
    };
};

void CudaSet::initialize(const size_t RecordCount, const unsigned int ColumnCount)
{
    mRecCount = RecordCount;
    hostRecCount = RecordCount;
    mColumnCount = ColumnCount;
    filtered = 0;
};


void CudaSet::initialize(queue<string> op_sel, const queue<string> op_sel_as)
{
    mRecCount = 0;
    mColumnCount = (unsigned int)op_sel.size();
    segCount = 1;
    not_compressed = 1;
    filtered = 0;
    col_aliases = op_sel_as;
    unsigned int i = 0;
    CudaSet *a;
    while(!op_sel.empty()) {
        for(auto it = varNames.begin(); it != varNames.end(); it++) {
            a = it->second;
            if(std::find(a->columnNames.begin(), a->columnNames.end(), op_sel.front()) != a->columnNames.end())
                break;
        };

        type[op_sel.front()] = a->type[op_sel.front()];
        cols[i] = op_sel.front();
        decimal[op_sel.front()] = a->decimal[op_sel.front()];
        columnNames.push_back(op_sel.front());

        if (a->type[op_sel.front()] == 0)  {
            d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
            //h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            h_columns_int[op_sel.front()] = thrust::host_vector<int_type>();
        }
        else if (a->type[op_sel.front()] == 1) {
            d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
            //h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            h_columns_float[op_sel.front()] = thrust::host_vector<float_type>();
        }
        else {
            h_columns_char[op_sel.front()] = nullptr;
            d_columns_char[op_sel.front()] = nullptr;
            char_size[op_sel.front()] = a->char_size[op_sel.front()];
        };
        i++;
        op_sel.pop();
    };
}



void CudaSet::initialize(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    mRecCount = 0;
    mColumnCount = 0;
    queue<string> q_cnt(op_sel);
    unsigned int i = 0;
    set<string> field_names;
    while(!q_cnt.empty()) {
        if( std::find(a->columnNames.begin(), a->columnNames.end(), q_cnt.front()) !=  a->columnNames.end() ||
                std::find(b->columnNames.begin(), b->columnNames.end(), q_cnt.front()) !=  b->columnNames.end())  {
            field_names.insert(q_cnt.front());
        };
        q_cnt.pop();
    }
    mColumnCount = (unsigned int)field_names.size();
    maxRecs = b->maxRecs;
    segCount = 1;
    filtered = 0;
    not_compressed = 1;

    col_aliases = op_sel_as;
    i = 0;
    while(!op_sel.empty()) {
        if(std::find(columnNames.begin(), columnNames.end(), op_sel.front()) ==  columnNames.end()) {
            if(std::find(a->columnNames.begin(), a->columnNames.end(), op_sel.front()) !=  a->columnNames.end()) {
                cols[i] = op_sel.front();
                decimal[op_sel.front()] = a->decimal[op_sel.front()];
                columnNames.push_back(op_sel.front());
                type[op_sel.front()] = a->type[op_sel.front()];

                if (a->type[op_sel.front()] == 0)  {
                    d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
                    h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    //h_columns_int[op_sel.front()] = thrust::host_vector<int_type>();
                    if(a->string_map.find(op_sel.front()) != a->string_map.end()) {
                        string_map[op_sel.front()] = a->string_map[op_sel.front()];
                        //cout << "SETTING J " << op_sel.front() << " " << a->string_map[op_sel.front()] << endl;
                    };
                }
                else if (a->type[op_sel.front()] == 1) {
                    d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
                    h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    //h_columns_float[op_sel.front()] = thrust::host_vector<float_type>();
                }
                else {
                    h_columns_char[op_sel.front()] = nullptr;
                    d_columns_char[op_sel.front()] = nullptr;
                    char_size[op_sel.front()] = a->char_size[op_sel.front()];
                    string_map[op_sel.front()] = a->string_map[op_sel.front()];
                    //cout << "SETTING J " << op_sel.front() << " " << a->string_map[op_sel.front()] << endl;
                };
                i++;
            }
            else if(std::find(b->columnNames.begin(), b->columnNames.end(), op_sel.front()) !=  b->columnNames.end()) {
                columnNames.push_back(op_sel.front());
                cols[i] = op_sel.front();
                decimal[op_sel.front()] = b->decimal[op_sel.front()];
                type[op_sel.front()] = b->type[op_sel.front()];

                if (b->type[op_sel.front()] == 0) {
                    d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
                    h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    //h_columns_int[op_sel.front()] = thrust::host_vector<int_type>();
                    if(b->string_map.find(op_sel.front()) != b->string_map.end()) {
                        string_map[op_sel.front()] = b->string_map[op_sel.front()];
                        //cout << "SETTING J " << op_sel.front() << " " << b->string_map[op_sel.front()] << endl;
                    };

                }
                else if (b->type[op_sel.front()] == 1) {
                    d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
                    h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    //h_columns_float[op_sel.front()] = thrust::host_vector<float_type>();
                }
                else {
                    h_columns_char[op_sel.front()] = nullptr;
                    d_columns_char[op_sel.front()] = nullptr;
                    char_size[op_sel.front()] = b->char_size[op_sel.front()];
                    string_map[op_sel.front()] = b->string_map[op_sel.front()];
                    //cout << "SETTING J " << op_sel.front() << " " << b->string_map[op_sel.front()] << endl;
                };
                i++;
            }
        }
        op_sel.pop();
    };
};



int_type reverse_op(int_type op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
    size_t available, total;
    hipMemGetInfo(&available, &total);
    return available;
} ;



void allocColumns(CudaSet* a, queue<string> fields)
{
    if(a->filtered) {
        CudaSet* t;
        if(a->filtered)
            t = varNames[a->source_name];
        else
            t = a;

        if(int_size*t->maxRecs > alloced_sz) {
            if(alloced_sz) {
                hipFree(alloced_tmp);
            };
            hipMalloc((void **) &alloced_tmp, int_size*t->maxRecs);
            alloced_sz = int_size*t->maxRecs;
        }
    }
    else {
        while(!fields.empty()) {
            if(var_exists(a, fields.front()) && !a->onDevice(fields.front())) {
                a->allocColumnOnDevice(fields.front(), a->maxRecs);
            }
            fields.pop();
        };
    };
}



void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment, size_t& count)
{
    if(!a->onDevice(field)) {
        a->allocColumnOnDevice(field, a->maxRecs);
    };
    if(a->prm_index == 'R') {
        mygather(field, a, t, count, a->mRecCount);
    }
    else {
        mycopy(field, a, t, count, t->mRecCount);
        a->mRecCount = t->mRecCount;
    };
}


size_t getSegmentRecCount(CudaSet* a, unsigned int segment) {
    if (segment == a->segCount-1) {
        return a->hostRecCount - a->maxRecs*segment;
    }
    else
        return 	a->maxRecs;
}

void copyFinalize(CudaSet* a, queue<string> fields)
{
	set<string> uniques;   
	if(scratch.size() < a->mRecCount*8)
		scratch.resize(a->mRecCount*8);
	thrust::device_ptr<int_type> tmp((int_type*)thrust::raw_pointer_cast(scratch.data()));		
   
	while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()) && cpy_bits.find(fields.front()) != cpy_bits.end())	{
						
			if(cpy_bits[fields.front()] == 8) {				
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<char> src((char*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));					
					thrust::transform(src, src+a->mRecCount, tmp, char_to_int64());					
				}
				else {
					thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, char_to_int64());
				};	
			}
			else if(cpy_bits[fields.front()] == 16) {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, int16_to_int64());
				}
				else {
					thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, int16_to_int64());
				};	
			}
			else if(cpy_bits[fields.front()] == 32) {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, int32_to_int64());
				}	
				else {
					thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, int32_to_int64());
				};
			}
			else {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<int_type> src((int_type*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::copy(src, src+a->mRecCount, tmp);
				}	
				else {					
					thrust::device_ptr<int_type> src((int_type*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::copy(src, src+a->mRecCount, tmp);
				};
			};			
			thrust::constant_iterator<int_type> iter(cpy_init_val[fields.front()]);
			if(a->type[fields.front()] != 1) {
				thrust::transform(tmp, tmp + a->mRecCount, iter, a->d_columns_int[fields.front()].begin(), thrust::plus<int_type>());				
			}
			else {
				thrust::device_ptr<int_type> dest((int_type*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
				thrust::transform(tmp, tmp + a->mRecCount, iter, dest, thrust::plus<int_type>());	
                thrust::transform(dest, dest+a->mRecCount, a->d_columns_float[fields.front()].begin(), long_to_float());				
			};				
		};		
		uniques.insert(fields.front());
        fields.pop();
    };   
}


void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt)
{
    set<string> uniques;
    if(a->filtered) { //filter the segment
        if(flt) {
            filter_op(a->fil_s, a->fil_f, segment);
        };
        if(rsz && a->mRecCount) {
            queue<string> fields1(fields);
            while(!fields1.empty()) {
                a->resizeDeviceColumn(a->devRecCount + a->mRecCount, fields1.front());
                fields1.pop();
            };
            a->devRecCount = a->devRecCount + a->mRecCount;
        };
    };
	cpy_bits.clear();
	cpy_init_val.clear();
	auto f(fields);
	
    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()))	{
            if(a->filtered) {
                if(a->mRecCount) {
                    CudaSet *t = varNames[a->source_name];
                    alloced_switch = 1;
                    t->CopyColumnToGpu(fields.front(), segment);
                    gatherColumns(a, t, fields.front(), segment, count);
                    alloced_switch = 0;
                    if(t->orig_segs.size() >= segment+1) {
                        a->orig_segs.resize(segment+1);
                        a->orig_segs.resize(segment+1);
                        a->orig_segs[segment] = t->orig_segs[segment];
                    };
                };
            }
            else {
                if(a->mRecCount) {
                    a->CopyColumnToGpu(fields.front(), segment, count);
                };
            };
            uniques.insert(fields.front());
        };
        fields.pop();
    };
}


void setPrm(CudaSet* a, CudaSet* b, char val, unsigned int segment)
{
    b->prm_index = val;
    if (val == 'A') {
        b->mRecCount = getSegmentRecCount(a,segment);
    }
    else if (val == 'N') {
        b->mRecCount = 0;
    }
}

void mygather(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[colname] != 1 ) {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<unsigned short int> d_col_source((unsigned short int*)alloced_tmp);
					thrust::device_ptr<unsigned short int> d_col_dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}	
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_int[colname].begin() + offset);
			};					
		}
		else {
			thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
			thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_int[colname].begin() + offset);			
		};

    }
    else  {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression			
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<unsigned short int> d_col_source((unsigned short int*)alloced_tmp);
					thrust::device_ptr<unsigned short int> d_col_dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}	
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_float[colname].begin() + offset);
			};					
		}
		else {		
			thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
			thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_float[colname].begin() + offset);
		};	
    }
};

void mycopy(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[colname] != 1) {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<short int> d_col_source((short int*)alloced_tmp);
					thrust::device_ptr<short int> d_col_dest((short int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()+offset));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}	
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col_source((int_type*)alloced_tmp);
					thrust::copy(d_col_source, d_col_source + g_size, a->d_columns_int[colname].begin() + offset);
			};					
		}
		else {		
			thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
			thrust::copy(d_col, d_col + g_size, a->d_columns_int[colname].begin() + offset);
		};
    }
    else {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
					thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
					thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 16) {
					thrust::device_ptr<short int> d_col_source((short int*)alloced_tmp);
					thrust::device_ptr<short int> d_col_dest((short int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()+offset));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else if(cpy_bits[colname] == 32) {
					thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
					thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);						
			}	
			else if(cpy_bits[colname] == 64) {
					thrust::device_ptr<int_type> d_col_source((int_type*)alloced_tmp);
					thrust::copy(d_col_source, d_col_source + g_size, a->d_columns_float[colname].begin() + offset);
			};					
		}
		else {		
			thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
			thrust::copy(d_col, d_col + g_size,	a->d_columns_float[colname].begin() + offset);
		};	
	};
};



size_t load_queue(queue<string> c1, CudaSet* right, string f2, size_t &rcount,
                  unsigned int start_segment, unsigned int end_segment, bool rsz, bool flt)
{
    queue<string> cc;
    while(!c1.empty()) {
        if(std::find(right->columnNames.begin(), right->columnNames.end(), c1.front()) !=  right->columnNames.end()) {
            if(f2 != c1.front() ) {
                cc.push(c1.front());
            };
        };
        c1.pop();
    };
    if(std::find(right->columnNames.begin(), right->columnNames.end(), f2) !=  right->columnNames.end()) {
        cc.push(f2);
    };

    if(right->filtered) {
        allocColumns(right, cc);
    };

    rcount = right->maxRecs;
    queue<string> ct(cc);

    while(!ct.empty()) {
        if(right->filtered && rsz) {
            right->mRecCount = 0;
        }
        else {
            right->allocColumnOnDevice(ct.front(), rcount);
        };
        ct.pop();
    };

    size_t cnt_r = 0;
    right->devRecCount = 0;
    for(unsigned int i = start_segment; i < end_segment; i++) {
        if(!right->filtered)
            copyColumns(right, cc, i, cnt_r, rsz, 0);
        else
            copyColumns(right, cc, i, cnt_r, rsz, flt);
        cnt_r = cnt_r + right->mRecCount;
    };

    right->mRecCount = cnt_r;
    return cnt_r;

}

size_t max_char(CudaSet* a)
{
    size_t max_char1 = 8;
    for(unsigned int i = 0; i < a->columnNames.size(); i++) {
        if(a->type[a->columnNames[i]] == 2) {
            if (a->char_size[a->columnNames[i]] > max_char1)
                max_char1 = a->char_size[a->columnNames[i]];
        }
        else if(a->type[a->columnNames[i]] == 0 && a->string_map.find(a->columnNames[i]) != a->string_map.end()) {
            auto s = a->string_map[a->columnNames[i]];
            auto pos = s.find_first_of(".");
            auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
            if (len > max_char1)
                max_char1 = len;
        };
    };
    return max_char1;
};


size_t max_char(CudaSet* a, queue<string> field_names)
{
    size_t max_char = 8;
    while (!field_names.empty()) {
        if (a->type[field_names.front()] == 2) {
            if (a->char_size[field_names.front()] > max_char)
                max_char = a->char_size[field_names.front()];
        };
        field_names.pop();
    };
    return max_char;
};


void setSegments(CudaSet* a, queue<string> cols)
{
    size_t mem_available = getFreeMem();
    size_t tot_sz = 0;
    while(!cols.empty()) {
        if(a->type[cols.front()] != 2)
            tot_sz = tot_sz + int_size;
        else
            tot_sz = tot_sz + a->char_size[cols.front()];
        cols.pop();
    };
    if(a->mRecCount*tot_sz > mem_available/3) { //default is 3
        a->segCount = (a->mRecCount*tot_sz)/(mem_available/5) + 1;
        a->maxRecs = (a->mRecCount/a->segCount)+1;
    };
};


void update_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);

    if (SortType.compare("DESC") == 0 )
        str_sort_host(tmp, RecCount, permutation, 1, len);
    else
        str_sort_host(tmp, RecCount, permutation, 0, len);
}


void apply_permutation_char(char* key, unsigned int* permutation, size_t RecCount, char* tmp, unsigned int len)
{
    // copy keys to temporary vector
    hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);
    // permute the keys
    str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, char* res, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}


void filter_op(const char *s, const char *f, unsigned int segment)
{
    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
    //std::clock_t start1 = std::clock();

    if(a->mRecCount == 0 && !a->filtered) {
        b = new CudaSet(0,1);
    }
    else {
        if(verbose)
            cout << "FILTER " << s << " " << f << " " << getFreeMem() << '\xd';

        b = varNames[s];
        b->name = s;
        b->string_map = a->string_map;
        size_t cnt = 0;
        allocColumns(a, b->fil_value);

        if (b->prm_d.size() == 0)
            b->prm_d.resize(a->maxRecs);

        //cout << endl << "MAP CHECK start " << segment <<  endl;
        char map_check = zone_map_check(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, a, segment);
        //cout << endl << "MAP CHECK segment " << segment << " " << map_check <<  endl;

        if(map_check == 'R') {
			auto old_ph = phase_copy;
			phase_copy = 0;
            copyColumns(a, b->fil_value, segment, cnt);
			phase_copy = old_ph;			
            bool* res = filter(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, a, segment);
            thrust::device_ptr<bool> bp((bool*)res);
            b->prm_index = 'R';
            b->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 1);
            thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
                            bp, b->prm_d.begin(), thrust::identity<bool>());
            hipFree(res);
        }
        else  {
            setPrm(a,b,map_check,segment);
        };
        if(segment == a->segCount-1)
            a->deAllocOnDevice();
    }
    if(verbose)
        cout << endl << "filter res " << b->mRecCount << " " << phase_copy << endl;
    //std::cout<< "filter time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
}




size_t load_right(CudaSet* right, string colname, string f2, queue<string> op_g, queue<string> op_sel,
                  queue<string> op_alt, bool decimal_join, size_t& rcount, unsigned int start_seg, unsigned int end_seg, bool rsz) {

    size_t cnt_r = 0;
    //if join is on strings then add integer columns to left and right tables and modify colInd1 and colInd2

    // need to allocate all right columns
    if(right->not_compressed) {
        queue<string> op_alt1;
        op_alt1.push(f2);
        cnt_r = load_queue(op_alt1, right, "", rcount, start_seg, end_seg, rsz, 1);
    }
    else {
        cnt_r = load_queue(op_alt, right, f2, rcount, start_seg, end_seg, rsz, 1);
    };


    /*    if (right->type[colname]  == 2) {
            str_join = 1;
            right->d_columns_int[f2] = thrust::device_vector<int_type>();
            for(unsigned int i = start_seg; i < end_seg; i++) {
                right->add_hashed_strings(f2, i);
            };
            cnt_r = right->d_columns_int[f2].size();
        };
    */
	
	
    if(right->not_compressed) {
        queue<string> op_alt1;
        while(!op_alt.empty()) {
            if(f2.compare(op_alt.front())) {
                if (std::find(right->columnNames.begin(), right->columnNames.end(), op_alt.front()) != right->columnNames.end()) {
                    op_alt1.push(op_alt.front());
                };
            };
            op_alt.pop();
        };
        if(!op_alt1.empty())
            cnt_r = load_queue(op_alt1, right, "", rcount, start_seg, end_seg, 0, 0);
    };
    return cnt_r;
};



void insert_records(const char* f, const char* s) {
    char buf[4096];
    size_t size, maxRecs, cnt = 0;
    string str_s, str_d;

    if(varNames.find(s) == varNames.end()) {
        process_error(3, "couldn't find " + string(s) );
    };
    CudaSet *a;
    a = varNames.find(s)->second;
    a->name = s;

    if(varNames.find(f) == varNames.end()) {
        process_error(3, "couldn't find " + string(f) );
    };

    CudaSet *b;
    b = varNames.find(f)->second;
    b->name = f;

    // if both source and destination are on disk
    cout << "SOURCES " << a->source << ":" << b->source << endl;
    if(a->source && b->source) {
        for(unsigned int i = 0; i < a->segCount; i++) {
            for(unsigned int z = 0; z < a->columnNames.size(); z++) {			
		
				if(a->type[a->columnNames[z]] != 2) {
					str_s = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i);
					str_d = b->load_file_name + "." + a->columnNames[z] + "." + to_string(b->segCount + i);
					cout << str_s << " " << str_d << endl;
					FILE* source = fopen(str_s.c_str(), "rb");
					FILE* dest = fopen(str_d.c_str(), "wb");
					while (size = fread(buf, 1, BUFSIZ, source)) {
						fwrite(buf, 1, size, dest);
					}
					fclose(source);
					fclose(dest);
				}
				else { //merge strings
					//read b's strings
					str_s = b->load_file_name + "." + b->columnNames[z];
					FILE* dest = fopen(str_s.c_str(), "rb");
					auto len = b->char_size[b->columnNames[z]];
					map<string, unsigned long long int> map_d;
					buf[len] = 0;
					unsigned long long cnt = 0;
					while (fread(buf, len, 1, dest)) {
						map_d[buf] = cnt;
						cnt++;
					};
					fclose(dest);
					unsigned long long int cct = cnt;
					
					str_s = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i) + ".hash";
					str_d = b->load_file_name + "." + b->columnNames[z] + "." + to_string(b->segCount + i) + ".hash";
					FILE* source = fopen(str_s.c_str(), "rb");
					dest = fopen(str_d.c_str(), "wb");
					while (size = fread(buf, 1, BUFSIZ, source)) {
						fwrite(buf, 1, size, dest);
					}
					fclose(source);
					fclose(dest);
					
					str_s = a->load_file_name + "." + a->columnNames[z];
					source = fopen(str_s.c_str(), "rb");
					map<unsigned long long int, string> map_s;
					buf[len] = 0;
					cnt = 0;
					while (fread(buf, len, 1, source)) {
						map_s[cnt] = buf;
						cnt++;
					};
					fclose(source);
					
					queue<string> op_vx;
					op_vx.push(a->columnNames[z]);
					allocColumns(a, op_vx);
					a->resize(a->maxRecs);
					a->CopyColumnToGpu(a->columnNames[z], z, 0);
					a->CopyColumnToHost(a->columnNames[z]);
					
					str_d = b->load_file_name + "." + b->columnNames[z];					
                    fstream f_file;
                    f_file.open(str_d.c_str(), ios::out|ios::app|ios::binary);
					
					for(auto j = 0; j < a->mRecCount; j++) {
						auto ss = map_s[a->h_columns_int[a->columnNames[z]][j]];
						if(map_d.find(ss) == map_d.end()) { //add
							f_file.write((char *)ss.c_str(), len);
							a->h_columns_int[a->columnNames[z]][j] = cct;
							cct++;							
						}
						else {
							a->h_columns_int[a->columnNames[z]][j] = map_d[ss];
						};
					};											
					f_file.close();	
					
					thrust::device_vector<int_type> d_col(a->mRecCount);
					thrust::copy(a->h_columns_int[a->columnNames[z]].begin(), a->h_columns_int[a->columnNames[z]].begin() + a->mRecCount, d_col.begin());
					auto i_name = b->load_file_name + "." + b->columnNames[z] + "." + to_string(b->segCount + i) + ".idx";
					pfor_compress(thrust::raw_pointer_cast(d_col.data()), a->mRecCount*int_size, i_name, a->h_columns_int[a->columnNames[z]], 0);					
				};	
            };
        };

        if(a->maxRecs > b->maxRecs)
            maxRecs = a->maxRecs;
        else
            maxRecs = b->maxRecs;

        for(unsigned int i = 0; i < b->columnNames.size(); i++) {
            b->reWriteHeader(b->load_file_name, b->columnNames[i], a->segCount + b->segCount, a->totalRecs + b->totalRecs, maxRecs);
        };
    }
    else if(!a->source && !b->source) { //if both source and destination are in memory
        size_t oldCount = b->mRecCount;
        b->resize(a->mRecCount);
        for(unsigned int z = 0; z< b->mColumnCount; z++) {
            if(b->type[a->columnNames[z]] == 0) {
                thrust::copy(a->h_columns_int[a->columnNames[z]].begin(), a->h_columns_int[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_int[b->columnNames[z]].begin() + oldCount);
            }
            else if(b->type[a->columnNames[z]] == 1) {
                thrust::copy(a->h_columns_float[a->columnNames[z]].begin(), a->h_columns_float[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_float[b->columnNames[z]].begin() + oldCount);
            }
            else {
                hipMemcpy(b->h_columns_char[b->columnNames[z]] + b->char_size[b->columnNames[z]]*oldCount, a->h_columns_char[a->columnNames[z]], a->char_size[a->columnNames[z]]*a->mRecCount, hipMemcpyHostToHost);
            };
        };
    }
    else if(!a->source && b->source) {

        total_segments = b->segCount;
        total_count = b->mRecCount;
        total_max = b->maxRecs;;

        queue<string> op_vx;
        for(unsigned int i=0; i < a->columnNames.size(); i++)
            op_vx.push(a->columnNames[i]);

        allocColumns(a, op_vx);
        a->resize(a->maxRecs);
        for(unsigned int i = 0; i < a->segCount; i++) {
            if (a->filtered) {
                copyColumns(a, op_vx, i, cnt);
                a->CopyToHost(0, a->mRecCount);
            };
            a->compress(b->load_file_name, 0, 1, i - (a->segCount-1), a->mRecCount);
        };
        for(unsigned int i = 0; i < b->columnNames.size(); i++) {
            b->writeHeader(b->load_file_name, b->columnNames[i], total_segments);
        };
    };
};



void delete_records(const char* f) {

    CudaSet *a;
    a = varNames.find(f)->second;
    a->name = f;
    size_t totalRemoved = 0;
    size_t maxRecs = 0;

    if(!a->keep) { // temporary variable
        process_error(2, "Delete operator is only applicable to disk based sets\nfor deleting records from derived sets please use filter operator ");
    }
    else {  // read matching segments, delete, compress and write on a disk replacing the original segments

        string str, str_old;
        queue<string> op_vx;
        size_t cnt;
        for ( auto it=data_dict[a->load_file_name].begin() ; it != data_dict[a->load_file_name].end(); ++it ) {
            op_vx.push((*it).first);
            if (std::find(a->columnNames.begin(), a->columnNames.end(), (*it).first) == a->columnNames.end()) {

                if ((*it).second.col_type == 0) {
                    a->type[(*it).first] = 0;
                    a->decimal[(*it).first] = 0;
                    //a->h_columns_int[(*it).first] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    a->h_columns_int[(*it).first] = thrust::host_vector<int_type>();
                    a->d_columns_int[(*it).first] = thrust::device_vector<int_type>();
                }
                else if((*it).second.col_type == 1) {
                    a->type[(*it).first] = 1;
                    a->decimal[(*it).first] = 0;
                    //a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    a->h_columns_float[(*it).first] = thrust::host_vector<float_type>();
                    a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
                }
                else if ((*it).second.col_type == 3) {
                    a->type[(*it).first] = 1;
                    a->decimal[(*it).first] = 1;
                    //a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    a->h_columns_float[(*it).first] = thrust::host_vector<float_type>();
                    a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
                }
                else {
                    a->type[(*it).first] = 2;
                    a->decimal[(*it).first] = 0;
                    a->h_columns_char[(*it).first] = nullptr;
                    a->d_columns_char[(*it).first] = nullptr;
                    a->char_size[(*it).first] = (*it).second.col_length;
                };
                a->columnNames.push_back((*it).first);
            }
        };

        allocColumns(a, op_vx);
        a->resize(a->maxRecs);
        a->prm_d.resize(a->maxRecs);
        size_t cc = a->mRecCount;
        size_t tmp;

        void* d;
        CUDA_SAFE_CALL(hipMalloc((void **) &d, a->maxRecs*float_size));
        unsigned int new_seg_count = 0;
        char map_check;

        for(unsigned int i = 0; i < a->segCount; i++) {

            map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, a, i);
            if(verbose)
                cout << "MAP CHECK segment " << i << " " << map_check <<  endl;
            if(map_check != 'N') {

                cnt = 0;
                copyColumns(a, op_vx, i, cnt);
                tmp = a->mRecCount;

                if(a->mRecCount) {
                    bool* res = filter(op_type,op_value,op_nums, op_nums_f, a, i);
                    thrust::device_ptr<bool> bp((bool*)res);
                    thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
                                    bp, a->prm_d.begin(), not_identity<bool>());

                    a->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 0);
                    hipFree(res);

//					cout << "Remained recs count " << a->mRecCount << endl;
                    if(a->mRecCount > maxRecs)
                        maxRecs = a->mRecCount;

                    if (a->mRecCount) {

                        totalRemoved = totalRemoved + (tmp - a->mRecCount);
                        if (a->mRecCount == tmp) { //none deleted
                            if(new_seg_count != i) {
                                for (auto it=data_dict[a->load_file_name].begin() ; it != data_dict[a->load_file_name].end(); ++it ) {
                                    auto colname = (*it).first;
                                    str_old = a->load_file_name + "." + colname + "." + to_string(i);
                                    str = a->load_file_name + "." + colname + "." + to_string(new_seg_count);
                                    remove(str.c_str());
                                    rename(str_old.c_str(), str.c_str());
                                };
                            };
                            new_seg_count++;

                        }
                        else { //some deleted
                            //cout << "writing segment " << new_seg_count << endl;

                            map<string, col_data> s = data_dict[a->load_file_name];
                            for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
                                string colname = (*it).first;
                                str = a->load_file_name + "." + colname + "." + to_string(new_seg_count);

                                if(a->type[colname] == 0) {
                                    thrust::device_ptr<int_type> d_col((int_type*)d);
                                    thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[colname].begin(), d_col);
                                    pfor_compress( d, a->mRecCount*int_size, str, a->h_columns_int[colname], 0);
                                }
                                else if(a->type[colname] == 1) {
                                    thrust::device_ptr<float_type> d_col((float_type*)d);
                                    if(a->decimal[colname]) {
                                        thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
                                        thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                                        thrust::transform(d_col,d_col+a->mRecCount, d_col_dec, float_to_long());
                                        pfor_compress( d, a->mRecCount*float_size, str, a->h_columns_float[colname], 1);
                                    }
                                    else {
                                        thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
                                        thrust::copy(d_col, d_col + a->mRecCount, a->h_columns_float[colname].begin());
                                        fstream binary_file(str.c_str(),ios::out|ios::binary);
                                        binary_file.write((char *)&a->mRecCount, 4);
                                        binary_file.write((char *)(a->h_columns_float[colname].data()),a->mRecCount*float_size);
                                        unsigned int comp_type = 3;
                                        binary_file.write((char *)&comp_type, 4);
                                        binary_file.close();

                                    };
                                }
                                else {
                                    thrust::device_ptr<int_type> d_col((int_type*)d);
                                    thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[colname].begin(), d_col);
                                    pfor_compress( d, a->mRecCount*int_size, str + ".hash", a->h_columns_int[colname], 0);
                                };
                            };
                            new_seg_count++;
                        };
                    }
                    else {
                        totalRemoved = totalRemoved + tmp;
                    };
                }
            }
            else {
                if(new_seg_count != i) {
                    for(unsigned int z = 0; z < a->columnNames.size(); z++) {
                        str_old = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i);
                        str = a->load_file_name + "." + a->columnNames[z] + "." + to_string(new_seg_count);
                        remove(str.c_str());
                        rename(str_old.c_str(), str.c_str());
                    };
                };
                new_seg_count++;
                maxRecs	= a->maxRecs;
            };
        };

        if (new_seg_count < a->segCount) {
            for(unsigned int i = new_seg_count; i < a->segCount; i++) {
                //cout << "delete segment " << i << endl;
                for(unsigned int z = 0; z < a->columnNames.size(); z++) {
                    str = a->load_file_name + "." + a->columnNames[z];
                    str += "." + to_string(i);
                    remove(str.c_str());
                };
            };
        };

        for(unsigned int i = new_seg_count; i < a->segCount; i++) {
            a->reWriteHeader(a->load_file_name, a->columnNames[i], new_seg_count, a->totalRecs-totalRemoved, maxRecs);
        };


        a->mRecCount = cc;
        a->prm_d.resize(0);
        a->segCount = new_seg_count;
        a->deAllocOnDevice();
        hipFree(d);
    };


};


void save_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
    size_t str_len;
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    size_t len = data_dict.size();
    binary_file.write((char *)&len, 8);
    for (auto it=data_dict.begin() ; it != data_dict.end(); ++it ) {
        str_len = (*it).first.size();
        binary_file.write((char *)&str_len, 8);
        binary_file.write((char *)(*it).first.data(), str_len);
        map<string, col_data> s = (*it).second;
        size_t len1 = s.size();
        binary_file.write((char *)&len1, 8);

        for (auto sit=s.begin() ; sit != s.end(); ++sit ) {
            str_len = (*sit).first.size();
            binary_file.write((char *)&str_len, 8);
            binary_file.write((char *)(*sit).first.data(), str_len);
            binary_file.write((char *)&(*sit).second.col_type, 4);
            binary_file.write((char *)&(*sit).second.col_length, 4);
        };
    };
    binary_file.close();
}

void load_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
    size_t str_len, recs, len1;
    string str1, str2;
    char buffer[4000];
    unsigned int col_type, col_length;
    fstream binary_file;
    binary_file.open(file_name.c_str(),ios::in|ios::binary);
    if(binary_file.is_open()) {
        binary_file.read((char*)&recs, 8);
        for(unsigned int i = 0; i < recs; i++) {
            binary_file.read((char*)&str_len, 8);
            binary_file.read(buffer, str_len);
            str1.assign(buffer, str_len);
            binary_file.read((char*)&len1, 8);

            for(unsigned int j = 0; j < len1; j++) {
                binary_file.read((char*)&str_len, 8);
                binary_file.read(buffer, str_len);
                str2.assign(buffer, str_len);
                binary_file.read((char*)&col_type, 4);
                binary_file.read((char*)&col_length, 4);
                data_dict[str1][str2].col_type = col_type;
                data_dict[str1][str2].col_length = col_length;
                //cout << "data DICT " << str1 << " " << str2 << " " << col_type << " " << col_length << endl;
            };
        };
        binary_file.close();
    }
    else {
        cout << "Couldn't open data dictionary" << endl;
    };
}

bool var_exists(CudaSet* a, string name) {

    if(std::find(a->columnNames.begin(), a->columnNames.end(), name) !=  a->columnNames.end())
        return 1;
    else

        return 0;
}

int file_exist (const char *filename)
{
    std::ifstream infile(filename);
    return infile.good();
}

bool check_bitmap_file_exist(CudaSet* left, CudaSet* right)
{
    queue<string> cols(right->fil_value);
    bool bitmaps_exist = 1;

    if(cols.size() == 0) {
        bitmaps_exist = 0;
    };
    while(cols.size() ) {
        if (std::find(right->columnNames.begin(), right->columnNames.end(), cols.front()) != right->columnNames.end()) {
            string fname = left->load_file_name + "."  + right->load_file_name + "." + cols.front() + ".0";
            if( !file_exist(fname.c_str())) {
                bitmaps_exist = 0;
            };
        };
        cols.pop();
    };
    return bitmaps_exist;
}

bool check_bitmaps_exist(CudaSet* left, CudaSet* right)
{
    //check if there are join bitmap indexes
    queue<string> cols(right->fil_value);
    bool bitmaps_exist = 1;

    if(cols.size() == 0) {
        bitmaps_exist = 1;
        return 1;
    };
    while(cols.size() ) {
        if (std::find(right->columnNames.begin(), right->columnNames.end(), cols.front()) != right->columnNames.end()) {
            string fname = left->load_file_name + "."  + right->load_file_name + "." + cols.front() + ".0";
            if( !file_exist(fname.c_str())) {
                bitmaps_exist = 0;
            };
        };
        cols.pop();
    };
    if(bitmaps_exist) {
        while(!right->fil_nums.empty() ) {
            left->fil_nums.push(right->fil_nums.front());
            right->fil_nums.pop();
        };
        while(!right->fil_nums_f.empty() ) {
            left->fil_nums_f.push(right->fil_nums_f.front());
            right->fil_nums_f.pop();
        };
        while(!right->fil_value.empty() ) {
            if (std::find(right->columnNames.begin(), right->columnNames.end(), right->fil_value.front()) != right->columnNames.end()) {
                string fname = left->load_file_name + "."  + right->load_file_name + "." + right->fil_value.front();
                left->fil_value.push(fname);
            }
            else
                left->fil_value.push(right->fil_value.front());
            right->fil_value.pop();
        };
        bool add_and = 1;
        if(left->fil_type.empty())
            add_and = 0;
        while(!right->fil_type.empty() ) {
            left->fil_type.push(right->fil_type.front());
            right->fil_type.pop();
        };
        if(add_and) {
            left->fil_type.push("AND");
        };
        return 1;
    }
    else {
        return 0;
    };
}


void check_sort(const string str, const char* rtable, const char* rid)
{
    CudaSet* right = varNames.find(rtable)->second;
    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::app);
    binary_file.write((char *)&right->sort_check, 1);
    binary_file.close();
}

void update_char_permutation(CudaSet* a, string colname, unsigned int* raw_ptr, string ord, void* temp, bool host)
{    
    auto s = a->string_map[colname];
    auto pos = s.find_first_of(".");
    auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
	
    a->h_columns_char[colname] = new char[a->mRecCount*len];
    memset(a->h_columns_char[colname], 0, a->mRecCount*len);
	
	thrust::device_ptr<unsigned int> perm(raw_ptr);
	thrust::device_ptr<int_type> temp_int((int_type*)temp);	
	thrust::gather(perm, perm+a->mRecCount, a->d_columns_int[colname].begin(), temp_int);
	
	//for(int z = 0 ; z < a->mRecCount; z++) {
	//cout << "Init vals " << a->d_columns_int[colname][z] << " " << perm[z] << " " << temp_int[z] << endl;
	//};
	
	//cout << "sz " << a->h_columns_int[colname].size() << " " << a->d_columns_int[colname].size() <<  " " << len << endl;
	hipMemcpy(thrust::raw_pointer_cast(a->h_columns_int[colname].data()), temp, 8*a->mRecCount, hipMemcpyDeviceToHost);

    FILE *f;
    f = fopen(a->string_map[colname].c_str(), "rb");

    for(int z = 0 ; z < a->mRecCount; z++) {
        fseek(f, a->h_columns_int[colname][z] * len, SEEK_SET);
        fread(a->h_columns_char[colname] + z*len, 1, len, f);
    };
    fclose(f);

    if(!host) {
        void *d;
        hipMalloc((void **) &d, a->mRecCount*len);
        a->d_columns_char[colname] = (char*)d;

        hipMemcpy(a->d_columns_char[colname], a->h_columns_char[colname], len*a->mRecCount, hipMemcpyHostToDevice);
		
	    if (ord.compare("DESC") == 0 )
			str_sort(a->d_columns_char[colname], a->mRecCount, raw_ptr, 1, len);
		else
			str_sort(a->d_columns_char[colname], a->mRecCount, raw_ptr, 0, len);
			
        hipFree(d);
    }
    else {
	    if (ord.compare("DESC") == 0 )
			str_sort_host(a->h_columns_char[colname], a->mRecCount, raw_ptr, 1, len);
		else
			str_sort_host(a->h_columns_char[colname], a->mRecCount, raw_ptr, 0, len);
    };
}



#ifdef _WIN64
size_t getTotalSystemMemory()
{
    MEMORYSTATUSEX status;
    status.dwLength = sizeof(status);
    GlobalMemoryStatusEx(&status);
    return status.ullTotalPhys;
}
#else
size_t getTotalSystemMemory()
{
    long pages = sysconf(_SC_PHYS_PAGES);
    long page_size = sysconf(_SC_PAGE_SIZE);
    return pages * page_size;
}
#endif

