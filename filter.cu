/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cm.h"

struct cmp_functor_dict
{
    const unsigned long long* source;
    bool *dest;
    const unsigned int *pars;

    cmp_functor_dict(const unsigned long long int* _source, bool * _dest,  const unsigned int * _pars):
        source(_source), dest(_dest), pars(_pars) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int idx = pars[0];
        unsigned int cmp = pars[1];
        unsigned int bits = ((unsigned int*)source)[1];
        unsigned int fit_count = ((unsigned int*)source)[0];
        unsigned int int_sz = 64;

        //find the source index
        unsigned int src_idx = i/fit_count;
        // find the exact location
        unsigned int src_loc = i%fit_count;
        //right shift the values
        unsigned int shifted = ((fit_count-src_loc)-1)*bits;
        unsigned long long int tmp = source[src_idx+2]  >> shifted;
        // set  the rest of bits to 0
        tmp	= tmp << (int_sz - bits);
        tmp	= tmp >> (int_sz - bits);
        //printf("COMP1 %llu %d \n", tmp, idx);
        if(cmp == 4) { // ==
            if(tmp == idx)
                dest[i] = 1;
            else
                dest[i] = 0;
        }
        else  { // !=
            if(tmp == idx)
                dest[i] = 0;
            else
                dest[i] = 1;
        };
    }
};


struct gpu_regex
{
    char  *source;
    char *pattern;
    bool * dest;
    const unsigned int *len;

    gpu_regex(char * _source,char * _pattern, bool * _dest,
              const unsigned int * _len):
        source(_source), pattern(_pattern), dest(_dest), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        bool star = 0;
        int j = 0;
        char* s;
        char* p;
        char* str = source + len[0]*i;
        char* pat = pattern;

loopStart:
        for (s = str, p = pat; j < len[0] && *s; ++s, ++p, ++j) {
            switch (*p) {
            case '?':
                if (*s == '.') goto starCheck;
                break;
            case '%':
                star = 1;
                str = s, pat = p;
                do {
                    ++pat;
                }
                while (*pat == '%');
                if (!*pat) {
                    dest[i] = 1;
                    return;
                }
                goto loopStart;
            default:
                if (*s != *p)
                    goto starCheck;
                break;
            } /* endswitch */
        } /* endfor */
        while (*p == '%') ++p;
        dest[i] = !*p;
        return;

starCheck:
        if (!star) {
            dest[i] = 0;
            return;
        };
        str++;
        j++;
        goto loopStart;
    }
};









bool* filter(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a,
             unsigned int segment)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
    stack<float_type*> exe_vectors_f;
    stack<int_type> exe_nums;
    stack<bool*> bool_vectors;
    stack<float_type> exe_nums_f;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    float_type n1_f, n2_f, res_f;


    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();
        //cout << endl << ss << endl;

        if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("VECTOR") == 0 || ss.compare("FLOAT") == 0
                || ss.compare("STRING") == 0 || ss.compare("FIELD") == 0) {


            if (ss.compare("NUMBER") == 0) {
                exe_nums.push(op_nums.front());
                op_nums.pop();
                exe_type.push(ss);
            }
            else if (ss.compare("NAME") == 0 || ss.compare("STRING") == 0) {
                exe_value.push(op_value.front());
                op_value.pop();
                exe_type.push(ss);
            }
            else if(ss.compare("FIELD") == 0) {
                size_t pos1 = op_value.front().find_first_of(".", 0);
                string tbl = op_value.front().substr(0,pos1);
                string field = op_value.front().substr(pos1+1, string::npos);
                op_value.pop();
                CudaSet *b = varNames.find(tbl)->second;
                if(b->type[field] == 0) {
                    auto val = b->h_columns_int[field][0];
                    exe_nums.push(val);
                    exe_type.push("NUMBER");
                }
                if(b->type[field] == 1) {
                    auto val = b->h_columns_float[field][0];
                    exe_nums_f.push(val);
                    exe_type.push("FLOAT");
                }
                else { // not for now

                };
            }
            else if (ss.compare("FLOAT") == 0) {
                exe_nums_f.push(op_nums_f.front());
                op_nums_f.pop();
                exe_type.push(ss);
            }

        }
        else {
            if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                // get 2 values from the stack

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();

                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();

                    if (ss.compare("ADD") == 0 )
                        res = n1+n2;
                    else if (ss.compare("MUL") == 0 )
                        res = n1*n2;
                    else if (ss.compare("DIV") == 0 )
                        res = n1/n2;
                    else
                        res = n1-n2;

                    thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
                    thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);

                    exe_type.push("VECTOR");
                    exe_vectors.push(thrust::raw_pointer_cast(p));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (ss.compare("ADD") == 0 )
                        res_f = n1_f+n2_f;
                    else if (ss.compare("MUL") == 0 )
                        res_f = n1_f*n2_f;
                    else if (ss.compare("DIV") == 0 )
                        res_f = n1_f/n2_f;
                    else
                        res_f = n1_f-n2_f;

                    thrust::device_ptr<float_type> p = thrust::device_malloc<float_type>(a->mRecCount);
                    thrust::sequence(p, p+(a->mRecCount),res_f,(float_type)0);

                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(thrust::raw_pointer_cast(p));

                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,1));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,1));
                    };

                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,0));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(t,(float_type)n1,ss,1));

                    }
                    else {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(t,n1,ss,1));
                    };
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(t,(float_type)n1,ss,0));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(t,n1,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t1 = a->get_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_int_by_name(s2_val);
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,t1,ss,0));
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t1,t,ss,0));
                        };
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t1,t,ss,0));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,t1,ss,0));
                        };
                    }
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0 ) && s2.compare("NAME") == 0) {

                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,s3,ss,0));
                            //free s3
                            hipFree(s3);

                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,t, ss,0));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0 ) && s1.compare("NAME") == 0) {

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,t,ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(s3,n1, ss,1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,(float_type)n1, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") || s2.compare("VECTOR F") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(s3,n1, ss,0));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,(float_type)n1, ss,0));
                        hipFree(s3);
                    }
                }

                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s4 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    exe_vectors.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,0));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
            }

            else if (ss.compare("CMP") == 0) {


                int_type cmp_type = op_nums.front();
                op_nums.pop();

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();

                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1,n2,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,n2_f,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NUMBER") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,float_type(n2),cmp_type));
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,float_type(n2),cmp_type));
                }

                else if ((s1.compare("STRING") == 0 && s2.compare("NAME") == 0) ||
                         (s1.compare("NAME") == 0 && s2.compare("STRING") == 0))
                {

                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                        s1.swap(s2);
                        s1_val.swap(s2_val);
                    };

                    void* d_res, *d_v;
                    if(cmp_type != 7)
                        hipMalloc((void **) &d_res, a->mRecCount);
                    else
                        hipMalloc((void **) &d_res, a->hostRecCount);
                    thrust::device_ptr<bool> dd_res((bool*)d_res);

                    hipMalloc((void **) &d_v, 8);
                    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                    thrust::counting_iterator<unsigned int> begin(0);

                    if(s2_val.find(".") != string::npos) { //bitmap index
                        auto pos1 = s2_val.find_first_of(".");
                        auto pos2 = s2_val.find_last_of(".");
                        auto set = s2_val.substr(pos1+1, (pos2-pos1)-1);
                        auto col = s2_val.substr(pos2+1);
                        auto len = data_dict[set][col].col_length;

                        while(s1_val.length() < len)
                            s1_val = s1_val + '\0';

                        auto s1_hash = MurmurHash64A(&s1_val, len, hash_seed)/2;

                        if(a->idx_dictionary_int[s2_val].find(s1_hash) != a->idx_dictionary_int[s2_val].end()) {
                            dd_v[0] = a->idx_dictionary_int[s2_val][s1_hash];
                            dd_v[1] = (unsigned int)cmp_type;
                            cmp_functor_dict ff(a->idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
                            thrust::for_each(begin, begin + a->mRecCount, ff);
                        }
                        else {
                            hipMemset(d_res,0,a->mRecCount);
                        }
                    }
                    else {

                        auto s = a->string_map[s2_val];
                        auto pos = s.find_first_of(".");
                        auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;

                        dd_v[0] = len;
                        dd_v[1] = (unsigned int)s1_val.length() + 1;

                        if(cmp_type != 7) {
                            thrust::device_vector<unsigned long long int> vv(1);
                            while(s1_val.length() < len) {
                                s1_val = s1_val + '\0';
                            };

                            vv[0] = MurmurHash64A(&s1_val[0], s1_val.length(), hash_seed)/2;

                            string f1 = a->load_file_name + "." + s2_val + "." + to_string(segment) + ".hash";
                            FILE* f = fopen(f1.c_str(), "rb" );
                            unsigned long long int* buff = new unsigned long long int[a->mRecCount];
                            unsigned int cnt;
                            fread(&cnt, 4, 1, f);
                            fread(buff, a->mRecCount*8, 1, f);
                            fclose(f);
                            thrust::device_vector<unsigned long long int> vals(a->mRecCount);
                            thrust::copy(buff, buff+a->mRecCount, vals.begin());
                            if(cmp_type == 4) //==
                                thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::equal_to<unsigned long long int>());
                            else if(cmp_type == 3) //!=
                                thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::not_equal_to<unsigned long long int>());
                            delete [] buff;

                            //auto cnt = thrust::count(dd_res, dd_res + a->mRecCount, 1);
                            //cout << endl << "R" << endl;
                        }
                        else {
                            if(a->map_like.find(s2_val) == a->map_like.end()) {
						
                                void* d_str;
                                hipMalloc((void **) &d_str, len);
                                hipMemset(d_str,0,len);
                                hipMemcpy( d_str, (void *) s1_val.c_str(), s1_val.length(), hipMemcpyHostToDevice);
								
                                string f1 = a->load_file_name + "." + s2_val;
                                FILE* f = fopen(f1.c_str(), "rb" );
                                fseek(f, 0, SEEK_END);
                                long fileSize = ftell(f);
                                fseek(f, 0, SEEK_SET);																
                                								
								unsigned int pieces = 1;
								if(fileSize > getFreeMem()/2)
									pieces = fileSize /(getFreeMem()/2) + 1;
								auto piece_sz = fileSize/pieces;
								ldiv_t ldivresult = ldiv(fileSize/pieces, len);		
								if(ldivresult.rem != 0)
									piece_sz = fileSize/pieces + (len - ldivresult.rem);										
								thrust::device_vector<char> dev(piece_sz);	
								char* buff = new char[piece_sz];
								a->map_res[s2_val] = thrust::device_vector<unsigned int>();
								for(auto i = 0; i < pieces; i++) {	
									
									if(i == pieces-1)
										piece_sz = fileSize - piece_sz*i;											
									fread(buff, piece_sz, 1, f);	
									hipMemcpy( thrust::raw_pointer_cast(dev.data()), (void*)buff, piece_sz, hipMemcpyHostToDevice);

									gpu_regex ff(thrust::raw_pointer_cast(dev.data()), (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
									thrust::for_each(begin, begin + piece_sz/len, ff);
									
									auto cnt = thrust::count(dd_res, dd_res + piece_sz/len, 1);
									auto offset = a->map_res[s2_val].size();
									a->map_res[s2_val].resize(a->map_res[s2_val].size() + cnt);
									thrust::copy_if(thrust::make_counting_iterator((unsigned int)(i*(piece_sz/len))), thrust::make_counting_iterator((unsigned int)((i+1)*(piece_sz/len))),
													dd_res, a->map_res[s2_val].begin() + offset, thrust::identity<bool>());
								};				
								
								fclose(f);
								delete [] buff;												
								hipFree(d_str);
                                thrust::sort(a->map_res[s2_val].begin(), a->map_res[s2_val].end());
                                a->map_like[s2_val] = 1;


                            };
                            // now lets calc the current segments's matches
                            hipMemset(d_res, 0, a->hostRecCount);
                            binary_search(a->map_res[s2_val].begin(),a->map_res[s2_val].end(), a->d_columns_int[s2_val].begin(), a->d_columns_int[s2_val].end(), dd_res);
                        };
                    };

                    hipFree(d_v);
                    exe_type.push("VECTOR");
                    bool_vectors.push((bool*)d_res);
                }

                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    //cout << "CMP " << s1_val << " " << n1 << " " << a->name << endl;

                    if(s1_val.find(".") != string::npos) { //bitmap index
                        void* d_v, *d_res;
                        hipMalloc((void **) &d_v, 8);
                        thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                        hipMalloc((void **) &d_res, a->mRecCount);
                        if(a->idx_dictionary_int[s1_val].find(n1) != a->idx_dictionary_int[s1_val].end()) {
                            dd_v[0] = a->idx_dictionary_int[s1_val][n1];
                            dd_v[1] = (unsigned int)cmp_type;

                            thrust::counting_iterator<unsigned int> begin(0);
                            cmp_functor_dict ff(a->idx_vals[s1_val], (bool*)d_res, (unsigned int*)d_v);
                            thrust::for_each(begin, begin + a->mRecCount, ff);
                        }
                        else {
                            hipMemset(d_res,0,a->mRecCount);
                        };
                        exe_type.push("VECTOR");
                        bool_vectors.push((bool*)d_res);
                        hipFree(d_v);
                    }
                    else {
                        if (a->type[s1_val] == 0) {
                            int_type* t = a->get_int_by_name(s1_val);
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,n1,cmp_type));
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,(float_type)n1,cmp_type));
                        };
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if(s2_val.find(".") != string::npos) { //bitmap index
                        void* d_v, *d_res;
                        hipMalloc((void **) &d_v, 8);
                        thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                        hipMalloc((void **) &d_res, a->mRecCount);

                        if(a->idx_dictionary_int[s2_val].find(n1) != a->idx_dictionary_int[s2_val].end()) {

                            dd_v[0] = a->idx_dictionary_int[s2_val][n1];
                            dd_v[1] = (unsigned int)cmp_type;

                            thrust::counting_iterator<unsigned int> begin(0);
                            cmp_functor_dict ff(a->idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
                            thrust::for_each(begin, begin + a->mRecCount, ff);
                        }
                        else {
                            hipMemset(d_res,0,a->mRecCount);
                        };
                        exe_type.push("VECTOR");
                        bool_vectors.push((bool*)d_res);
                        hipFree(d_v);
                    }
                    else {
                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_int_by_name(s2_val);
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,n1,cmp_type));
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,(float_type)n1,cmp_type));
                        };
                    };
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(int_type)n1_f,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1_f,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(int_type)n1_f,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1_f,cmp_type));
                    };
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR F") == 0) {

                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") == 0) {

                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("VECTOR") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(int_type)n1_f,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(int_type)n1_f,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NAME") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }


                else if (s1.compare("VECTOR") == 0 && s2.compare("NAME") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR F") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s2,s3,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s2 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s2,s3,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }


                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s1_val] == 0 || a->type[s1_val] == 2) {
                        int_type* t = a->get_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            bool_vectors.push(a->compare(t1,t,cmp_type));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            bool_vectors.push(a->compare(t1,t,cmp_type));
                        };
                    }
                    else {
                        cmp_type = reverse_op(cmp_type);
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            bool_vectors.push(a->compare(t,t1,cmp_type));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            bool_vectors.push(a->compare(t,t1,cmp_type));
                        };
                    }
                }
            }

            else if (ss.compare("AND") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(a->logical_and(s2,s3));
            }
            else if (ss.compare("OR") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(a->logical_or(s2,s3));
            }
            else {
                cout << "found nothing " << endl;
            }
        };
    };

    return bool_vectors.top();
}